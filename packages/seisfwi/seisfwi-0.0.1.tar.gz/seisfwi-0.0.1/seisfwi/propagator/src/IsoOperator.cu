#include "Boundary.h"
#include "Cpml.h"
#include "Model.h"
#include "Parameter.h"
#include "Survey.h"
#include "utils.h"
#include <chrono>
#include <string>
using std::string;


extern "C" void Propagator(const string para_fname, const int gpu_id,
                           const int group_size, const int *shot_ids,
                           const float *Lambda, const float *Mu,
                           const float *Den, const float *stf,
                           const bool is_acoustic, const bool with_adj) {

  // Set GPU device
  CHECK(hipSetDevice(gpu_id));
  auto start0 = std::chrono::high_resolution_clock::now();

  // Read parameter file
  Parameter para(para_fname);
  int nz = para.nz();
  int nx = para.nx();
  int nt = para.nt();
  int npml = para.npml();
  int npad = para.npad();
  float dx = para.dx();
  float dz = para.dz();
  float dt = para.dt();
  float f0 = para.f0();
  float weight_pr = para.weight_pr();
  float weight_vx = para.weight_vx();
  float weight_vz = para.weight_vz();
  float weight_et = para.weight_et();

  // Set default values
  int nrec = 1;
  int ndas = 1;

  // Transpose models and convert to float
  float *fLambda, *fMu, *fDen;
  fLambda = (float *)malloc(nz * nx * sizeof(float));
  fMu = (float *)malloc(nz * nx * sizeof(float));
  fDen = (float *)malloc(nz * nx * sizeof(float));
  for (int i = 0; i < nz; i++) {
    for (int j = 0; j < nx; j++) {
      fLambda[j * nz + i] = Lambda[i * nx + j] * MEGA;
      fMu[j * nz + i] = Mu[i * nx + j] * MEGA;
      fDen[j * nz + i] = Den[i * nx + j];
    }
  }

  // Set up model
  Model model(para, fLambda, fMu, fDen);

  // Set up CPML boundary conditions
  Cpml cpml(para, model);

  // Set up source and receiver
  bool with_residual = false;
  Survey survey(para, with_adj, with_residual, stf, group_size, shot_ids);

  // Compute Courant number
  compCourantNumber(model.h_Vp, nz * nx, dt, dz, dx);

  // Set up GPU threads and blocks
  dim3 threads(TX, TY);
  dim3 blocks((nz + TX - 1) / TX, (nx + TY - 1) / TY);

  // Define device memory
  float *d_vz, *d_vx, *d_szz, *d_sxx, *d_sxz;
  float *d_mem_dvz_dz, *d_mem_dvz_dx, *d_mem_dvx_dz, *d_mem_dvx_dx;
  float *d_mem_dszz_dz, *d_mem_dsxx_dx, *d_mem_dsxz_dz, *d_mem_dsxz_dx;
  float *d_gauss_amp;

  float *d_syn_pr; // pressure data
  float *d_syn_vx; // vertical velocity data
  float *d_syn_vz; // horizontal velocity data
  float *d_syn_et; // tangential strain data
  float *d_adj_pr; // pressure, as I use explosive source

  // Allocate device memory: forward wavefield
  CHECK(hipMalloc((void **)&d_vz, nz * nx * sizeof(float)));
  CHECK(hipMalloc((void **)&d_vx, nz * nx * sizeof(float)));
  CHECK(hipMalloc((void **)&d_szz, nz * nx * sizeof(float)));
  CHECK(hipMalloc((void **)&d_sxx, nz * nx * sizeof(float)));
  CHECK(hipMalloc((void **)&d_sxz, nz * nx * sizeof(float)));

  // Allocate device memory: memory variables for PML
  CHECK(hipMalloc((void **)&d_mem_dvz_dz, nz * nx * sizeof(float)));
  CHECK(hipMalloc((void **)&d_mem_dvz_dx, nz * nx * sizeof(float)));
  CHECK(hipMalloc((void **)&d_mem_dvx_dz, nz * nx * sizeof(float)));
  CHECK(hipMalloc((void **)&d_mem_dvx_dx, nz * nx * sizeof(float)));
  CHECK(hipMalloc((void **)&d_mem_dszz_dz, nz * nx * sizeof(float)));
  CHECK(hipMalloc((void **)&d_mem_dsxx_dx, nz * nx * sizeof(float)));
  CHECK(hipMalloc((void **)&d_mem_dsxz_dz, nz * nx * sizeof(float)));
  CHECK(hipMalloc((void **)&d_mem_dsxz_dx, nz * nx * sizeof(float)));

  // Set the source gaussian amplitude
  CHECK(hipMalloc((void **)&d_gauss_amp, 81 * sizeof(float)));
  src_rec_gauss_amp<<<1, threads>>>(d_gauss_amp, 9, 9);

  // float *h_snap;
  // h_snap = (float *)malloc(nz * nx * sizeof(float));

  // Start the stream
  hipStream_t streams[group_size];

  // Modeling over shots
  for (int iShot = 0; iShot < group_size; iShot++) {
    // printf("  Processing shot %d\n", shot_ids[iShot]);

    // Set up the source
    CHECK(hipStreamCreate(&streams[iShot]));

    // Initialize the wavefields
    intialArrayGPU<<<blocks, threads>>>(d_vz, nz, nx, 0.0);
    intialArrayGPU<<<blocks, threads>>>(d_vx, nz, nx, 0.0);
    intialArrayGPU<<<blocks, threads>>>(d_szz, nz, nx, 0.0);
    intialArrayGPU<<<blocks, threads>>>(d_sxx, nz, nx, 0.0);
    intialArrayGPU<<<blocks, threads>>>(d_sxz, nz, nx, 0.0);

    intialArrayGPU<<<blocks, threads>>>(d_mem_dvz_dz, nz, nx, 0.0);
    intialArrayGPU<<<blocks, threads>>>(d_mem_dvz_dx, nz, nx, 0.0);
    intialArrayGPU<<<blocks, threads>>>(d_mem_dvx_dz, nz, nx, 0.0);
    intialArrayGPU<<<blocks, threads>>>(d_mem_dvx_dx, nz, nx, 0.0);
    intialArrayGPU<<<blocks, threads>>>(d_mem_dszz_dz, nz, nx, 0.0);
    intialArrayGPU<<<blocks, threads>>>(d_mem_dsxx_dx, nz, nx, 0.0);
    intialArrayGPU<<<blocks, threads>>>(d_mem_dsxz_dz, nz, nx, 0.0);
    intialArrayGPU<<<blocks, threads>>>(d_mem_dsxz_dx, nz, nx, 0.0);

    // Read the receiver data
    nrec = survey.vec_nrec.at(iShot);
    ndas = survey.vec_ndas.at(iShot);

    // Allocate device memory for syn data and intialize to zero
    CHECK(hipMalloc((void **)&d_syn_pr, nrec * nt * sizeof(float)));
    CHECK(hipMalloc((void **)&d_syn_vx, nrec * nt * sizeof(float)));
    CHECK(hipMalloc((void **)&d_syn_vz, nrec * nt * sizeof(float)));
    CHECK(hipMalloc((void **)&d_syn_et, ndas * nt * sizeof(float)));
    intialArrayGPU<<<blocks, threads>>>(d_syn_pr, nt, nrec, 0.0);
    intialArrayGPU<<<blocks, threads>>>(d_syn_vx, nt, nrec, 0.0);
    intialArrayGPU<<<blocks, threads>>>(d_syn_vz, nt, nrec, 0.0);
    intialArrayGPU<<<blocks, threads>>>(d_syn_et, nt, ndas, 0.0);

    // ------------------------ forward time loop (elastic) ----------------------------
    if (is_acoustic) {

      // std::cout << "Forward modeling: acoustic" << std::endl;

      for (int it = 0; it <= nt - 2; it++) {
        // Update the stress
        acoustic_forward_pressure<<<blocks, threads>>>(
            d_vz, d_vx, d_szz, d_mem_dvz_dz, d_mem_dvx_dx, model.d_Lambda,
            cpml.d_K_z_half, cpml.d_a_z_half, cpml.d_b_z_half, cpml.d_K_x,
            cpml.d_a_x, cpml.d_b_x, nz, nx, dt, dz, dx, npml, npad);

        // Add explosive source
        add_source<<<1, 1>>>(d_szz, d_sxx, survey.vec_source.at(iShot)[it], nz,
                             survey.vec_z_src.at(iShot),
                             survey.vec_x_src.at(iShot), dt, d_gauss_amp);

        // Update the velocity
        acoustic_forward_velocity<<<blocks, threads>>>(
            d_vz, d_vx, d_szz, d_mem_dszz_dz, d_mem_dsxx_dx, model.d_ave_Byc_a,
            model.d_ave_Byc_b, cpml.d_K_z, cpml.d_a_z, cpml.d_b_z,
            cpml.d_K_x_half, cpml.d_a_x_half, cpml.d_b_x_half, nz, nx, dt, dz,
            dx, npml, npad);

        // Record syn data recored by geophone: pr = sxx + szz, vx, vz
        record_geo<<<(nrec + 31) / 32, 32>>>(
            d_szz, d_sxx, d_vx, d_vz, nz, d_syn_pr, d_syn_vx, d_syn_vz,
            iShot, it + 1, nt, nrec, survey.d_vec_z_rec.at(iShot),
            survey.d_vec_x_rec.at(iShot));

        // Record syn data recored by DAS: et
        record_das<<<(ndas + 31) / 32, 32>>>(
          d_vx, d_vz, nz, d_syn_et,
          iShot, it + 1, nt, ndas, survey.d_vec_z_das.at(iShot),
          survey.d_vec_x_das.at(iShot), survey.d_vec_das_wt_x.at(iShot), 
          survey.d_vec_das_wt_z.at(iShot), survey.gl);
      }
    } else {
      // std::cout << "Forward modeling: elastic" << std::endl;
      for (int it = 0; it <= nt - 2; it++) {

        // Update stress
        elastic_forward_stress<<<blocks, threads>>>(
            d_vz, d_vx, d_szz, d_sxx, d_sxz, d_mem_dvz_dz, d_mem_dvz_dx,
            d_mem_dvx_dz, d_mem_dvx_dx, model.d_Lambda, model.d_Mu,
            model.d_ave_Mu, cpml.d_K_z, cpml.d_a_z, cpml.d_b_z, cpml.d_K_z_half,
            cpml.d_a_z_half, cpml.d_b_z_half, cpml.d_K_x, cpml.d_a_x,
            cpml.d_b_x, cpml.d_K_x_half, cpml.d_a_x_half, cpml.d_b_x_half, nz,
            nx, dt, dz, dx, npml, npad);

        // Add explosive source
        add_source<<<1, threads>>>(d_szz, d_sxx,
                                   survey.vec_source.at(iShot)[it], nz,
                                   survey.vec_z_src.at(iShot),
                                   survey.vec_x_src.at(iShot), dt, d_gauss_amp);

        // Update velocity
        elastic_forward_velocity<<<blocks, threads>>>(
            d_vz, d_vx, d_szz, d_sxx, d_sxz, d_mem_dszz_dz, d_mem_dsxz_dx,
            d_mem_dsxz_dz, d_mem_dsxx_dx, model.d_ave_Byc_a, model.d_ave_Byc_b,
            cpml.d_K_z, cpml.d_a_z, cpml.d_b_z, cpml.d_K_z_half,
            cpml.d_a_z_half, cpml.d_b_z_half, cpml.d_K_x, cpml.d_a_x,
            cpml.d_b_x, cpml.d_K_x_half, cpml.d_a_x_half, cpml.d_b_x_half, nz,
            nx, dt, dz, dx, npml, npad);

        // Record syn data recored by geophone: pr = sxx + szz, vx, vz
        record_geo<<<(nrec + 31) / 32, 32>>>(
            d_szz, d_sxx, d_vx, d_vz, nz, d_syn_pr, d_syn_vx, d_syn_vz,
            iShot, it + 1, nt, nrec, survey.d_vec_z_rec.at(iShot),
            survey.d_vec_x_rec.at(iShot));
        
        // Record syn data recored by DAS: et
        record_das<<<(ndas + 31) / 32, 32>>>(
          d_vx, d_vz, nz, d_syn_et,
          iShot, it + 1, nt, ndas, survey.d_vec_z_das.at(iShot),
          survey.d_vec_x_das.at(iShot), survey.d_vec_das_wt_x.at(iShot), 
          survey.d_vec_das_wt_z.at(iShot), survey.gl);

        // // Save wavefield for debug
        // if (iShot == 0 && it % 100 == 0) {
        //   CHECK(hipMemcpy(h_snap, d_szz, nz * nx * sizeof(float), hipMemcpyDeviceToHost));
        //   fileBinWrite(h_snap, nz * nx, "SnapGPU_" + std::to_string(it) + ".bin");
        // }

      } // end of forward time loop
    }
    // Copy data back to host memory, only for forward modeling
    CHECK(hipMemcpyAsync(survey.vec_syn_pr.at(iShot), d_syn_pr,
                          nt * nrec * sizeof(float), hipMemcpyDeviceToHost,
                          streams[iShot]));
    CHECK(hipMemcpyAsync(survey.vec_syn_vx.at(iShot), d_syn_vx,
                          nt * nrec * sizeof(float), hipMemcpyDeviceToHost,
                          streams[iShot]));
    CHECK(hipMemcpyAsync(survey.vec_syn_vz.at(iShot), d_syn_vz,
                          nt * nrec * sizeof(float), hipMemcpyDeviceToHost,
                          streams[iShot]));
    CHECK(hipMemcpyAsync(survey.vec_syn_et.at(iShot), d_syn_et,
                          nt * ndas * sizeof(float), hipMemcpyDeviceToHost,
                          streams[iShot]));

    // synchronize all streams
    hipDeviceSynchronize();

    if (with_adj) {
      // Allocate device memory for adj data and intialize to zero
      CHECK(hipMalloc((void **)&d_adj_pr, 1 * nt * sizeof(float)));
      intialArrayGPU<<<blocks, threads>>>(d_adj_pr, nt, 1, 0.0);

      //  Initialize the wavefields
      intialArrayGPU<<<blocks, threads>>>(d_vz, nz, nx, 0.0);
      intialArrayGPU<<<blocks, threads>>>(d_vx, nz, nx, 0.0);
      intialArrayGPU<<<blocks, threads>>>(d_szz, nz, nx, 0.0);
      intialArrayGPU<<<blocks, threads>>>(d_sxx, nz, nx, 0.0);
      intialArrayGPU<<<blocks, threads>>>(d_sxz, nz, nx, 0.0);

      intialArrayGPU<<<blocks, threads>>>(d_mem_dvz_dz, nz, nx, 0.0);
      intialArrayGPU<<<blocks, threads>>>(d_mem_dvz_dx, nz, nx, 0.0);
      intialArrayGPU<<<blocks, threads>>>(d_mem_dvx_dz, nz, nx, 0.0);
      intialArrayGPU<<<blocks, threads>>>(d_mem_dvx_dx, nz, nx, 0.0);
      intialArrayGPU<<<blocks, threads>>>(d_mem_dszz_dz, nz, nx, 0.0);
      intialArrayGPU<<<blocks, threads>>>(d_mem_dsxx_dx, nz, nx, 0.0);
      intialArrayGPU<<<blocks, threads>>>(d_mem_dsxz_dz, nz, nx, 0.0);
      intialArrayGPU<<<blocks, threads>>>(d_mem_dsxz_dx, nz, nx, 0.0);

      if (is_acoustic) {
        // std::cout << "Adjoint modeling: acoustic" << std::endl;
        for (int it = nt - 2; it >= 0; it--) {
          // update velocity of the adjoint wavefield
          acoustic_adjoint_velocity<<<blocks, threads>>>(
              d_vz, d_vx, d_szz, d_mem_dvz_dz, d_mem_dvx_dx, d_mem_dszz_dz,
              d_mem_dsxx_dx, model.d_Lambda, model.d_ave_Byc_a,
              model.d_ave_Byc_b, cpml.d_K_z_half, cpml.d_a_z_half,
              cpml.d_b_z_half, cpml.d_K_x_half, cpml.d_a_x_half,
              cpml.d_b_x_half, cpml.d_K_z, cpml.d_a_z, cpml.d_b_z, cpml.d_K_x,
              cpml.d_a_x, cpml.d_b_x, nz, nx, dt, dz, dx, npml, npad);

          // record pressure data
          record_adj<<<(nrec + 31) / 32, 32>>>(
              d_szz, d_sxx, nz, d_adj_pr, iShot, it, nt, 1,
              survey.vec_z_src.at(iShot), survey.vec_x_src.at(iShot));

          // inject geophone residuals
          inject_geo<<<(nrec + 31) / 32, 32>>>(
            d_szz, d_sxx, d_vx, d_vz, nz, d_syn_pr, d_syn_vx,
            d_syn_vz, it, dt, nt, nrec, survey.d_vec_z_rec.at(iShot), 
            survey.d_vec_x_rec.at(iShot), weight_pr, weight_vx, weight_vz);
        
          // inject das residuals
          inject_das<<<(ndas + 31) / 32, 32>>>(d_vx, d_vz, nz, d_syn_et, 
              it, dt, nt, ndas, survey.d_vec_z_das.at(iShot), 
              survey.d_vec_x_das.at(iShot), weight_et, 
              survey.d_vec_das_wt_x.at(iShot), 
              survey.d_vec_das_wt_z.at(iShot), survey.gl);

          // update stress of the adjoint wavefield
          acoustic_adjoint_pressure<<<blocks, threads>>>(
              d_vz, d_vx, d_szz, d_mem_dvz_dz, d_mem_dvx_dx, d_mem_dszz_dz,
              d_mem_dsxx_dx, model.d_Lambda, model.d_ave_Byc_a,
              model.d_ave_Byc_b, cpml.d_K_z_half, cpml.d_a_z_half,
              cpml.d_b_z_half, cpml.d_K_x_half, cpml.d_a_x_half,
              cpml.d_b_x_half, cpml.d_K_z, cpml.d_a_z, cpml.d_b_z, cpml.d_K_x,
              cpml.d_a_x, cpml.d_b_x, nz, nx, dt, dz, dx, npml, npad);
        }
      } else {
        // std::cout << "Adjoint modeling: elastic" << std::endl;
        // ----------adjoint time loop (elastic) ----------
        for (int it = nt - 2; it >= 0; it--) {

          // update velocity of the adjoint wavefield
          elastic_adjoint_velocity<<<blocks, threads>>>(
              d_vz, d_vx, d_szz, d_sxx, d_sxz, d_mem_dszz_dz, d_mem_dsxz_dx,
              d_mem_dsxz_dz, d_mem_dsxx_dx, d_mem_dvz_dz, d_mem_dvz_dx,
              d_mem_dvx_dz, d_mem_dvx_dx, model.d_Lambda, model.d_Mu,
              model.d_ave_Mu, model.d_Den, model.d_ave_Byc_a, model.d_ave_Byc_b,
              cpml.d_K_z_half, cpml.d_a_z_half, cpml.d_b_z_half,
              cpml.d_K_x_half, cpml.d_a_x_half, cpml.d_b_x_half, cpml.d_K_z,
              cpml.d_a_z, cpml.d_b_z, cpml.d_K_x, cpml.d_a_x, cpml.d_b_x, nz,
              nx, dt, dz, dx, npml, npad);

          // record pressure data
          record_adj<<<(nrec + 31) / 32, 32>>>(
              d_szz, d_sxx, nz, d_adj_pr, iShot, it, nt, 1,
              survey.vec_z_src.at(iShot), survey.vec_x_src.at(iShot));

          // inject geophone residuals
          inject_geo<<<(nrec + 31) / 32, 32>>>(
            d_szz, d_sxx, d_vx, d_vz, nz, d_syn_pr, d_syn_vx,
            d_syn_vz, it, dt, nt, nrec, survey.d_vec_z_rec.at(iShot), 
            survey.d_vec_x_rec.at(iShot), weight_pr, weight_vx, weight_vz);
        
          // inject das residuals
          inject_das<<<(ndas + 31) / 32, 32>>>(d_vx, d_vz, nz, d_syn_et, 
              it, dt, nt, ndas, survey.d_vec_z_das.at(iShot), 
              survey.d_vec_x_das.at(iShot), weight_et, 
              survey.d_vec_das_wt_x.at(iShot), 
              survey.d_vec_das_wt_z.at(iShot), survey.gl);

          // update velocity of the adjoint wavefield
          elastic_adjoint_stress<<<blocks, threads>>>(
              d_vz, d_vx, d_szz, d_sxx, d_sxz, d_mem_dszz_dz, d_mem_dsxz_dx,
              d_mem_dsxz_dz, d_mem_dsxx_dx, d_mem_dvz_dz, d_mem_dvz_dx,
              d_mem_dvx_dz, d_mem_dvx_dx, model.d_Lambda, model.d_Mu,
              model.d_ave_Mu, model.d_Den, model.d_ave_Byc_a, model.d_ave_Byc_b,
              cpml.d_K_z_half, cpml.d_a_z_half, cpml.d_b_z_half,
              cpml.d_K_x_half, cpml.d_a_x_half, cpml.d_b_x_half, cpml.d_K_z,
              cpml.d_a_z, cpml.d_b_z, cpml.d_K_x, cpml.d_a_x, cpml.d_b_x, nz,
              nx, dt, dz, dx, npml, npad);

        } // the end of backward time loop
      }

      // transfer the adjoint pressure data to cpu
      CHECK(hipMemcpyAsync(survey.vec_adj_pr.at(iShot), d_adj_pr,
                            nt * 1 * sizeof(float), hipMemcpyDeviceToHost,
                            streams[iShot]));

      // free the memory on device
      CHECK(hipFree(d_adj_pr));
    }

    // free memory on GPU
    CHECK(hipFree(d_syn_pr));
    CHECK(hipFree(d_syn_vx));
    CHECK(hipFree(d_syn_vz));
    CHECK(hipFree(d_syn_et));

  } // the end of shot loop

  // save the shot record
  for (int iShot = 0; iShot < group_size; iShot++) {
    fileBinWrite(survey.vec_syn_pr.at(iShot), nt * survey.vec_nrec.at(iShot),
                 para.data_dir_name() + "/shot" +
                     std::to_string(shot_ids[iShot]) + "_pr.bin");
    fileBinWrite(survey.vec_syn_vx.at(iShot), nt * survey.vec_nrec.at(iShot),
                 para.data_dir_name() + "/shot" +
                     std::to_string(shot_ids[iShot]) + "_vx.bin");
    fileBinWrite(survey.vec_syn_vz.at(iShot), nt * survey.vec_nrec.at(iShot),
                 para.data_dir_name() + "/shot" +
                     std::to_string(shot_ids[iShot]) + "_vz.bin");
    fileBinWrite(survey.vec_syn_et.at(iShot), nt * survey.vec_ndas.at(iShot),
                 para.data_dir_name() + "/shot" +
                     std::to_string(shot_ids[iShot]) + "_et.bin");

    if (with_adj) {
      fileBinWrite(survey.vec_adj_pr.at(iShot), nt * 1,
                   para.data_dir_name() + "/adj_shot" +
                       std::to_string(shot_ids[iShot]) + "_pr.bin");
    }
  }

  free(fLambda);
  free(fMu);
  free(fDen);
  // free(h_snap);

  // destroy the streams
  for (int iShot = 0; iShot < group_size; iShot++) {
    CHECK(hipStreamDestroy(streams[iShot]));
  }

  hipFree(d_vz);
  hipFree(d_vx);
  hipFree(d_szz);
  hipFree(d_sxx);
  hipFree(d_sxz);
  hipFree(d_mem_dvz_dz);
  hipFree(d_mem_dvz_dx);
  hipFree(d_mem_dvx_dz);
  hipFree(d_mem_dvx_dx);
  hipFree(d_mem_dszz_dz);
  hipFree(d_mem_dsxx_dx);
  hipFree(d_mem_dsxz_dz);
  hipFree(d_mem_dsxz_dx);
  hipFree(d_gauss_amp);
}



extern "C" void Gradient(const string para_fname, const int gpu_id,
                         const int group_size, const int *shot_ids,
                         const float *Lambda, const float *Mu, const float *Den,
                         const float *stf, float *misfit, float *grad_Lambda,
                         float *grad_Mu, float *grad_Den, float *grad_Vp,
                         float *grad_stf, const bool is_acoustic,
                         const bool with_grad) {

  // Set GPU device
  CHECK(hipSetDevice(gpu_id));
  auto start0 = std::chrono::high_resolution_clock::now();

  // Read parameter file
  Parameter para(para_fname);
  int nz = para.nz();
  int nx = para.nx();
  int nt = para.nt();
  int npml = para.npml();
  int npad = para.npad();
  float dx = para.dx();
  float dz = para.dz();
  float dt = para.dt();
  float f0 = para.f0();
  float weight_pr = para.weight_pr();
  float weight_vx = para.weight_vx();
  float weight_vz = para.weight_vz();
  float weight_et = para.weight_et();

  // Set default values
  int nrec = 1;
  int ndas = 1;
  float win_ratio = 0.000; // 0.005
  float amp_ratio = 1.0;

  // Transpose models and convert to float
  float *fLambda, *fMu, *fDen;
  fLambda = (float *)malloc(nz * nx * sizeof(float));
  fMu = (float *)malloc(nz * nx * sizeof(float));
  fDen = (float *)malloc(nz * nx * sizeof(float));
  for (int i = 0; i < nz; i++) {
    for (int j = 0; j < nx; j++) {
      fLambda[j * nz + i] = Lambda[i * nx + j] * MEGA;
      fMu[j * nz + i] = Mu[i * nx + j] * MEGA;
      fDen[j * nz + i] = Den[i * nx + j];
    }
  }

  // Set up model
  Model model(para, fLambda, fMu, fDen);

  // Set up CPML boundary conditions
  Cpml cpml(para, model);

  // Set up boundaries for reconstruction in gradient calculation
  Bnd boundaries(para, with_grad);

  // Set up source and receiver
  bool with_adj = false;
  bool with_residual = true;
  Survey survey(para, with_adj, with_residual, stf, group_size, shot_ids);

  // Compute Courant number
  compCourantNumber(model.h_Vp, nz * nx, dt, dz, dx);

  // Set up GPU threads and blocks
  dim3 threads(TX, TY);
  dim3 blocks((nz + TX - 1) / TX, (nx + TY - 1) / TY);

  // Define device memory
  float *d_vz, *d_vx, *d_szz, *d_sxx, *d_sxz;
  float *d_vz_adj, *d_vx_adj, *d_szz_adj, *d_sxx_adj, *d_sxz_adj;
  float *d_mem_dvz_dz, *d_mem_dvz_dx, *d_mem_dvx_dz, *d_mem_dvx_dx;
  float *d_mem_dszz_dz, *d_mem_dsxx_dx, *d_mem_dsxz_dz, *d_mem_dsxz_dx;
  float *d_obj_pr, *d_obj_vx, *d_obj_vz, *d_obj_et;
  float *d_gauss_amp;
  float *h_obj_pr = nullptr;
  float *h_obj_vx = nullptr;
  float *h_obj_vz = nullptr;
  float *h_obj_et = nullptr;
  float h_obj = 0.0;

  // for acoustic
  float *d_szz_p1;

  // Synthetic data
  float *d_syn_pr; // pressure
  float *d_syn_vx; // vertical velocity
  float *d_syn_vz; // horizontal velocity
  float *d_syn_et; // tangential strain

  // Observed data
  float *d_obs_pr; // pressure
  float *d_obs_vx; // vertical velocity
  float *d_obs_vz; // horizontal velocity
  float *d_obs_et; // tangential strain

  // Residual data
  float *d_res_pr; // pressure
  float *d_res_vx; // vertical velocity
  float *d_res_vz; // horizontal velocity
  float *d_res_et; // tangential strain

  // Allocate device memory: forward wavefield
  CHECK(hipMalloc((void **)&d_vz, nz * nx * sizeof(float)));
  CHECK(hipMalloc((void **)&d_vx, nz * nx * sizeof(float)));
  CHECK(hipMalloc((void **)&d_szz, nz * nx * sizeof(float)));
  CHECK(hipMalloc((void **)&d_sxx, nz * nx * sizeof(float)));
  CHECK(hipMalloc((void **)&d_sxz, nz * nx * sizeof(float)));

  // Allocate device memory: memory variables for computing gradient
  CHECK(hipMalloc((void **)&d_mem_dvz_dz, nz * nx * sizeof(float)));
  CHECK(hipMalloc((void **)&d_mem_dvz_dx, nz * nx * sizeof(float)));
  CHECK(hipMalloc((void **)&d_mem_dvx_dz, nz * nx * sizeof(float)));
  CHECK(hipMalloc((void **)&d_mem_dvx_dx, nz * nx * sizeof(float)));
  CHECK(hipMalloc((void **)&d_mem_dszz_dz, nz * nx * sizeof(float)));
  CHECK(hipMalloc((void **)&d_mem_dsxx_dx, nz * nx * sizeof(float)));
  CHECK(hipMalloc((void **)&d_mem_dsxz_dz, nz * nx * sizeof(float)));
  CHECK(hipMalloc((void **)&d_mem_dsxz_dx, nz * nx * sizeof(float)));

  // Allocate device memory: adjoint wavefield
  CHECK(hipMalloc((void **)&d_vz_adj, nz * nx * sizeof(float)));
  CHECK(hipMalloc((void **)&d_vx_adj, nz * nx * sizeof(float)));
  CHECK(hipMalloc((void **)&d_szz_adj, nz * nx * sizeof(float)));
  CHECK(hipMalloc((void **)&d_sxx_adj, nz * nx * sizeof(float)));
  CHECK(hipMalloc((void **)&d_sxz_adj, nz * nx * sizeof(float)));
  if (is_acoustic) {
    CHECK(hipMalloc((void **)&d_szz_p1, nz * nx * sizeof(float)));
  }
  // Allocate device memory: objective function
  CHECK(hipMalloc((void **)&d_obj_pr, 1 * sizeof(float)));
  CHECK(hipMalloc((void **)&d_obj_vx, 1 * sizeof(float)));
  CHECK(hipMalloc((void **)&d_obj_vz, 1 * sizeof(float)));
  CHECK(hipMalloc((void **)&d_obj_et, 1 * sizeof(float)));
  h_obj_pr = (float *)malloc(sizeof(float));
  h_obj_vx = (float *)malloc(sizeof(float));
  h_obj_vz = (float *)malloc(sizeof(float));
  h_obj_et = (float *)malloc(sizeof(float));

  // Set the source gaussian amplitude
  CHECK(hipMalloc((void **)&d_gauss_amp, 81 * sizeof(float)));
  src_rec_gauss_amp<<<1, threads>>>(d_gauss_amp, 9, 9);

 // float *h_snap;
 // h_snap = (float *)malloc(nz * nx * sizeof(float));

  // Start the stream
  hipStream_t streams[group_size];

  // Modeling over shots
  for (int iShot = 0; iShot < group_size; iShot++) {
    // printf("  Processing shot %d\n", shot_ids[iShot]);

    // Set grid and block size for residuals
    dim3 blocksT_rec((nt + TX - 1) / TX, (nrec + TY - 1) / TY);
    dim3 blocksT_das((nt + TX - 1) / TX, (ndas + TY - 1) / TY);

    // Set up the source
    CHECK(hipStreamCreate(&streams[iShot]));

    // Read the receiver data
    nrec = survey.vec_nrec.at(iShot);
    ndas = survey.vec_ndas.at(iShot);
      
    // Load the observed data
    fileBinLoad(survey.vec_obs_pr.at(iShot), nt * nrec,
                para.data_dir_name() + "/shot" +
                    std::to_string(shot_ids[iShot]) + "_pr.bin");
    fileBinLoad(survey.vec_obs_vx.at(iShot), nt * nrec,
                para.data_dir_name() + "/shot" +
                    std::to_string(shot_ids[iShot]) + "_vx.bin");
    fileBinLoad(survey.vec_obs_vz.at(iShot), nt * nrec,
                para.data_dir_name() + "/shot" +
                    std::to_string(shot_ids[iShot]) + "_vz.bin");
    fileBinLoad(survey.vec_obs_et.at(iShot), nt * ndas,
                para.data_dir_name() + "/shot" +
                    std::to_string(shot_ids[iShot]) + "_et.bin");

    // Allocate device memory for obs data and then copy from host to device
    CHECK(hipMalloc((void **)&d_obs_pr, nrec * nt * sizeof(float)));
    CHECK(hipMalloc((void **)&d_obs_vx, nrec * nt * sizeof(float)));
    CHECK(hipMalloc((void **)&d_obs_vz, nrec * nt * sizeof(float)));
    CHECK(hipMalloc((void **)&d_obs_et, ndas * nt * sizeof(float)));

    intialArrayGPU<<<blocksT_rec, threads>>>(d_obs_pr, nt, nrec, 0.0);
    intialArrayGPU<<<blocksT_rec, threads>>>(d_obs_vx, nt, nrec, 0.0);
    intialArrayGPU<<<blocksT_rec, threads>>>(d_obs_vz, nt, nrec, 0.0);
    intialArrayGPU<<<blocksT_das, threads>>>(d_obs_et, nt, ndas, 0.0);

    CHECK(hipMemcpyAsync(d_obs_pr, survey.vec_obs_pr.at(iShot), nrec * nt * sizeof(float), hipMemcpyHostToDevice,
                          streams[iShot]));
    CHECK(hipMemcpyAsync(d_obs_vx, survey.vec_obs_vx.at(iShot), nrec * nt * sizeof(float), hipMemcpyHostToDevice,
                          streams[iShot]));
    CHECK(hipMemcpyAsync(d_obs_vz, survey.vec_obs_vz.at(iShot), nrec * nt * sizeof(float), hipMemcpyHostToDevice,
                          streams[iShot]));
    CHECK(hipMemcpyAsync(d_obs_et, survey.vec_obs_et.at(iShot), ndas * nt * sizeof(float), hipMemcpyHostToDevice,
                          streams[iShot]));

    // Allocate device memory for syn data and intialize to zero
    CHECK(hipMalloc((void **)&d_syn_pr, nrec * nt * sizeof(float)));
    CHECK(hipMalloc((void **)&d_syn_vx, nrec * nt * sizeof(float)));
    CHECK(hipMalloc((void **)&d_syn_vz, nrec * nt * sizeof(float)));
    CHECK(hipMalloc((void **)&d_syn_et, ndas * nt * sizeof(float)));

    intialArrayGPU<<<blocksT_rec, threads>>>(d_syn_pr, nt, nrec, 0.0);
    intialArrayGPU<<<blocksT_rec, threads>>>(d_syn_vx, nt, nrec, 0.0);
    intialArrayGPU<<<blocksT_rec, threads>>>(d_syn_vz, nt, nrec, 0.0);
    intialArrayGPU<<<blocksT_das, threads>>>(d_syn_et, nt, ndas, 0.0);

    // Allocate device memory for res data and initialize to zero
    CHECK(hipMalloc((void **)&d_res_pr, nrec * nt * sizeof(float)));
    CHECK(hipMalloc((void **)&d_res_vx, nrec * nt * sizeof(float)));
    CHECK(hipMalloc((void **)&d_res_vz, nrec * nt * sizeof(float)));
    CHECK(hipMalloc((void **)&d_res_et, ndas * nt * sizeof(float)));

    intialArrayGPU<<<blocksT_rec, threads>>>(d_res_pr, nt, nrec, 0.0);
    intialArrayGPU<<<blocksT_rec, threads>>>(d_res_vx, nt, nrec, 0.0);
    intialArrayGPU<<<blocksT_rec, threads>>>(d_res_vz, nt, nrec, 0.0);
    intialArrayGPU<<<blocksT_das, threads>>>(d_res_et, nt, ndas, 0.0);

    // Initialize the wavefields for modeling
    intialArrayGPU<<<blocks, threads>>>(d_vz, nz, nx, 0.0);
    intialArrayGPU<<<blocks, threads>>>(d_vx, nz, nx, 0.0);
    intialArrayGPU<<<blocks, threads>>>(d_szz, nz, nx, 0.0);
    intialArrayGPU<<<blocks, threads>>>(d_sxx, nz, nx, 0.0);
    intialArrayGPU<<<blocks, threads>>>(d_sxz, nz, nx, 0.0);

    intialArrayGPU<<<blocks, threads>>>(d_mem_dvz_dz, nz, nx, 0.0);
    intialArrayGPU<<<blocks, threads>>>(d_mem_dvz_dx, nz, nx, 0.0);
    intialArrayGPU<<<blocks, threads>>>(d_mem_dvx_dz, nz, nx, 0.0);
    intialArrayGPU<<<blocks, threads>>>(d_mem_dvx_dx, nz, nx, 0.0);
    intialArrayGPU<<<blocks, threads>>>(d_mem_dszz_dz, nz, nx, 0.0);
    intialArrayGPU<<<blocks, threads>>>(d_mem_dsxx_dx, nz, nx, 0.0);
    intialArrayGPU<<<blocks, threads>>>(d_mem_dsxz_dz, nz, nx, 0.0);
    intialArrayGPU<<<blocks, threads>>>(d_mem_dsxz_dx, nz, nx, 0.0);


    if (is_acoustic) {

      // std::cout << "Forward modeling: acoustic" << std::endl;

      for (int it = 0; it <= nt - 2; it++) {

        // Save and record from the beginning, only for gradient calculation
        if (with_grad) {
          boundaries.field_from_bnd(d_szz, d_sxz, d_sxx, d_vz, d_vx, it);
        }

        // Update the stress
        acoustic_forward_pressure<<<blocks, threads>>>(
            d_vz, d_vx, d_szz, d_mem_dvz_dz, d_mem_dvx_dx, model.d_Lambda,
            cpml.d_K_z_half, cpml.d_a_z_half, cpml.d_b_z_half, cpml.d_K_x,
            cpml.d_a_x, cpml.d_b_x, nz, nx, dt, dz, dx, npml, npad);

        // Add explosive source
        add_source<<<1, threads>>>(d_szz, d_sxx,
                                   survey.vec_source.at(iShot)[it], nz,
                                   survey.vec_z_src.at(iShot),
                                   survey.vec_x_src.at(iShot), dt, d_gauss_amp);

        // Update the velocity
        acoustic_forward_velocity<<<blocks, threads>>>(
            d_vz, d_vx, d_szz, d_mem_dszz_dz, d_mem_dsxx_dx, model.d_ave_Byc_a,
            model.d_ave_Byc_b, cpml.d_K_z, cpml.d_a_z, cpml.d_b_z,
            cpml.d_K_x_half, cpml.d_a_x_half, cpml.d_b_x_half, nz, nx, dt, dz,
            dx, npml, npad);

        // Record syn data recored by geophone: pr = sxx + szz, vx, vz
        record_geo<<<(nrec + 31) / 32, 32>>>(
          d_szz, d_sxx, d_vx, d_vz, nz, d_syn_pr, d_syn_vx, d_syn_vz,
          iShot, it + 1, nt, nrec, survey.d_vec_z_rec.at(iShot),
          survey.d_vec_x_rec.at(iShot));
      
        // Record syn data recored by DAS: et
        record_das<<<(ndas + 31) / 32, 32>>>(
          d_vx, d_vz, nz, d_syn_et,
          iShot, it + 1, nt, ndas, survey.d_vec_z_das.at(iShot),
          survey.d_vec_x_das.at(iShot), survey.d_vec_das_wt_x.at(iShot), 
          survey.d_vec_das_wt_z.at(iShot),survey.gl);
      }
    } else {
      // std::cout << "Forward modeling: elastic" << std::endl;

      // Model the synthetic data: time loop (elastic)
      for (int it = 0; it <= nt - 2; it++) {

        // Save and record from the beginning, only for gradient calculation
        if (with_grad) {
          boundaries.field_from_bnd(d_szz, d_sxz, d_sxx, d_vz, d_vx, it);
        }

        // Update stress
        elastic_forward_stress<<<blocks, threads>>>(
            d_vz, d_vx, d_szz, d_sxx, d_sxz, d_mem_dvz_dz, d_mem_dvz_dx,
            d_mem_dvx_dz, d_mem_dvx_dx, model.d_Lambda, model.d_Mu,
            model.d_ave_Mu, cpml.d_K_z, cpml.d_a_z, cpml.d_b_z, cpml.d_K_z_half,
            cpml.d_a_z_half, cpml.d_b_z_half, cpml.d_K_x, cpml.d_a_x,
            cpml.d_b_x, cpml.d_K_x_half, cpml.d_a_x_half, cpml.d_b_x_half, nz,
            nx, dt, dz, dx, npml, npad);

        // Add explosive source
        add_source<<<1, threads>>>(d_szz, d_sxx,
                                   survey.vec_source.at(iShot)[it], nz,
                                   survey.vec_z_src.at(iShot),
                                   survey.vec_x_src.at(iShot), dt, d_gauss_amp);

        // Update velocity
        elastic_forward_velocity<<<blocks, threads>>>(
            d_vz, d_vx, d_szz, d_sxx, d_sxz, d_mem_dszz_dz, d_mem_dsxz_dx,
            d_mem_dsxz_dz, d_mem_dsxx_dx, model.d_ave_Byc_a, model.d_ave_Byc_b,
            cpml.d_K_z, cpml.d_a_z, cpml.d_b_z, cpml.d_K_z_half,
            cpml.d_a_z_half, cpml.d_b_z_half, cpml.d_K_x, cpml.d_a_x,
            cpml.d_b_x, cpml.d_K_x_half, cpml.d_a_x_half, cpml.d_b_x_half, nz,
            nx, dt, dz, dx, npml, npad);

        // Record syn data recored by geophone: pr = sxx + szz, vx, vz
        record_geo<<<(nrec + 31) / 32, 32>>>(
          d_szz, d_sxx, d_vx, d_vz, nz, d_syn_pr, d_syn_vx, d_syn_vz,
          iShot, it + 1, nt, nrec, survey.d_vec_z_rec.at(iShot),
          survey.d_vec_x_rec.at(iShot));
      
        // Record syn data recored by DAS: et
        record_das<<<(ndas + 31) / 32, 32>>>(
          d_vx, d_vz, nz, d_syn_et,
          iShot, it + 1, nt, ndas, survey.d_vec_z_das.at(iShot),
          survey.d_vec_x_das.at(iShot), survey.d_vec_das_wt_x.at(iShot), 
          survey.d_vec_das_wt_z.at(iShot), survey.gl);

      } // end of forward time loop
    }

    // ------------------------ Compute residuals ------------------------ //

    // Windowing the obs and syn data: pr. vx, vz, and et
    // if (para.if_win()) {
    //   cuda_window<<<blocksT_rec, threads>>>(
    //       nt, nrec, dt, survey.d_vec_win_start.at(iShot),
    //       survey.d_vec_win_end.at(iShot), win_ratio, d_obs_pr);
    //   cuda_window<<<blocksT_rec, threads>>>(
    //       nt, nrec, dt, survey.d_vec_win_start.at(iShot),
    //       survey.d_vec_win_end.at(iShot), win_ratio, d_syn_pr);
    //   cuda_window<<<blocksT_rec, threads>>>(
    //       nt, nrec, dt, survey.d_vec_win_start.at(iShot),
    //       survey.d_vec_win_end.at(iShot), win_ratio, d_obs_vx);
    //   cuda_window<<<blocksT_rec, threads>>>(
    //       nt, nrec, dt, survey.d_vec_win_start.at(iShot),
    //       survey.d_vec_win_end.at(iShot), win_ratio, d_syn_vx);
    //   cuda_window<<<blocksT_rec, threads>>>(
    //       nt, nrec, dt, survey.d_vec_win_start.at(iShot),
    //       survey.d_vec_win_end.at(iShot), win_ratio, d_obs_vz);
    //   cuda_window<<<blocksT_rec, threads>>>(
    //       nt, nrec, dt, survey.d_vec_win_start.at(iShot),
    //       survey.d_vec_win_end.at(iShot), win_ratio, d_syn_vz);
    //   cuda_window<<<blocksT_das, threads>>>(
    //       nt, ndas, dt, survey.d_vec_win_start.at(iShot),
    //       survey.d_vec_win_end.at(iShot), win_ratio, d_obs_et);
    //   cuda_window<<<blocksT_das, threads>>>(
    //       nt, ndas, dt, survey.d_vec_win_start.at(iShot),
    //       survey.d_vec_win_end.at(iShot), win_ratio, d_syn_et);

    // } else {
    //   cuda_window<<<blocksT_rec, threads>>>(nt, nrec, dt, win_ratio, d_obs_pr);
    //   cuda_window<<<blocksT_rec, threads>>>(nt, nrec, dt, win_ratio, d_syn_pr);
    //   cuda_window<<<blocksT_rec, threads>>>(nt, nrec, dt, win_ratio, d_obs_vx);
    //   cuda_window<<<blocksT_rec, threads>>>(nt, nrec, dt, win_ratio, d_syn_vx);
    //   cuda_window<<<blocksT_rec, threads>>>(nt, nrec, dt, win_ratio, d_obs_vz);
    //   cuda_window<<<blocksT_rec, threads>>>(nt, nrec, dt, win_ratio, d_syn_vz);
    //   cuda_window<<<blocksT_das, threads>>>(nt, ndas, dt, win_ratio, d_obs_et);
    //   cuda_window<<<blocksT_das, threads>>>(nt, ndas, dt, win_ratio, d_syn_et);
    // }


    // Calculate source update and filter calculated data
    if (para.if_src_update()) {
      amp_ratio = source_update(nt, dt, nrec, d_obs_pr, d_syn_pr,
                                survey.d_vec_source.at(iShot), survey.d_coef);
      printf("	Source update => Processing shot %d, amp_ratio = %f\n", iShot, amp_ratio);
    }
    amp_ratio = 1.0; // amplitude not used, so set to 1.0

    // TODO: change here to implement different objective functions
    //  objective function

    gpuMinus<<<blocksT_rec, threads>>>(d_res_pr, d_obs_pr, d_syn_pr, nt, nrec);
    gpuMinus<<<blocksT_rec, threads>>>(d_res_vx, d_obs_vx, d_syn_vx, nt, nrec);
    gpuMinus<<<blocksT_rec, threads>>>(d_res_vz, d_obs_vz, d_syn_vz, nt, nrec);
    gpuMinus<<<blocksT_das, threads>>>(d_res_et, d_obs_et, d_syn_et, nt, ndas);

    cuda_cal_objective<<<1, 512>>>(d_obj_pr, d_res_pr, nt * nrec);
    cuda_cal_objective<<<1, 512>>>(d_obj_vx, d_res_vx, nt * nrec);
    cuda_cal_objective<<<1, 512>>>(d_obj_vz, d_res_vz, nt * nrec);
    cuda_cal_objective<<<1, 512>>>(d_obj_et, d_res_et, nt * ndas);

    CHECK(hipMemcpy(h_obj_pr, d_obj_pr, sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(h_obj_vx, d_obj_vx, sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(h_obj_vz, d_obj_vz, sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(h_obj_et, d_obj_et, sizeof(float), hipMemcpyDeviceToHost));

    // Calculate objective function
    h_obj += h_obj_pr[0] * weight_pr + h_obj_vx[0] * weight_vx +
             h_obj_vz[0] * weight_vz + h_obj_et[0] * weight_et;

    //  update source again (adjoint)
    if (para.if_src_update()) {
      source_update_adj(nt, dt, nrec, d_res_pr, amp_ratio, survey.d_coef);
    }

    // // windowing again (adjoint)
    // if (para.if_win()) {
    //   cuda_window<<<blocksT_rec, threads>>>(
    //       nt, nrec, dt, survey.d_vec_win_start.at(iShot),
    //       survey.d_vec_win_end.at(iShot), win_ratio, d_res_pr);
    //   cuda_window<<<blocksT_rec, threads>>>(
    //       nt, nrec, dt, survey.d_vec_win_start.at(iShot),
    //       survey.d_vec_win_end.at(iShot), win_ratio, d_res_vx);
    //   cuda_window<<<blocksT_rec, threads>>>(
    //       nt, nrec, dt, survey.d_vec_win_start.at(iShot),
    //       survey.d_vec_win_end.at(iShot), win_ratio, d_res_vz);
    //   cuda_window<<<blocksT_das, threads>>>(
    //       nt, ndas, dt, survey.d_vec_win_start.at(iShot),
    //       survey.d_vec_win_end.at(iShot), win_ratio, d_res_et);
    // } else {
    //   cuda_window<<<blocksT_rec, threads>>>(nt, nrec, dt, win_ratio, d_res_pr);
    //   cuda_window<<<blocksT_rec, threads>>>(nt, nrec, dt, win_ratio, d_res_vx);
    //   cuda_window<<<blocksT_rec, threads>>>(nt, nrec, dt, win_ratio, d_res_vz);
    //   cuda_window<<<blocksT_das, threads>>>(nt, ndas, dt, win_ratio, d_res_et);
    // }


    if (para.if_save_scratch()){

      // copy the residual data from device to host (processed)
      CHECK(hipMemcpyAsync(survey.vec_res_pr.at(iShot), d_res_pr,
          nt * nrec * sizeof(float), hipMemcpyDeviceToHost,
          streams[iShot]));
      CHECK(hipMemcpyAsync(survey.vec_res_vx.at(iShot), d_res_vx,
          nt * nrec * sizeof(float), hipMemcpyDeviceToHost,
          streams[iShot]));
      CHECK(hipMemcpyAsync(survey.vec_res_vz.at(iShot), d_res_vz,
          nt * nrec * sizeof(float), hipMemcpyDeviceToHost,
          streams[iShot]));
      CHECK(hipMemcpyAsync(survey.vec_res_et.at(iShot), d_res_et,
          nt * ndas * sizeof(float), hipMemcpyDeviceToHost,
          streams[iShot]));

      // copy the synthetic data from device to host (processed)
      CHECK(hipMemcpyAsync(survey.vec_syn_pr.at(iShot), d_syn_pr,
                            nt * nrec * sizeof(float), hipMemcpyDeviceToHost,
                            streams[iShot]));
      CHECK(hipMemcpyAsync(survey.vec_syn_vx.at(iShot), d_syn_vx,
                            nt * nrec * sizeof(float), hipMemcpyDeviceToHost,
                            streams[iShot]));
      CHECK(hipMemcpyAsync(survey.vec_syn_vz.at(iShot), d_syn_vz,
                            nt * nrec * sizeof(float), hipMemcpyDeviceToHost,
                            streams[iShot]));
      CHECK(hipMemcpyAsync(survey.vec_syn_et.at(iShot), d_syn_et,
                            nt * ndas * sizeof(float), hipMemcpyDeviceToHost,
                            streams[iShot]));

      // copy the observed data from device to host (processed)
      CHECK(hipMemcpyAsync(survey.vec_obs_pr.at(iShot), d_obs_pr,
                            nt * nrec * sizeof(float), hipMemcpyDeviceToHost,
                            streams[iShot]));
      CHECK(hipMemcpyAsync(survey.vec_obs_vx.at(iShot), d_obs_vx,
                            nt * nrec * sizeof(float), hipMemcpyDeviceToHost,
                            streams[iShot]));
      CHECK(hipMemcpyAsync(survey.vec_obs_vz.at(iShot), d_obs_vz,
                            nt * nrec * sizeof(float), hipMemcpyDeviceToHost,
                            streams[iShot]));
      CHECK(hipMemcpyAsync(survey.vec_obs_et.at(iShot), d_obs_et,
                            nt * ndas * sizeof(float), hipMemcpyDeviceToHost,
                            streams[iShot]));
    }

    // copy the source wavelet from device to host
    CHECK(hipMemcpy(survey.vec_source.at(iShot), survey.d_vec_source.at(iShot),
                     nt * sizeof(float), hipMemcpyDeviceToHost));

    // ------------------------ Compute residuals ------------------------ //

    // synchronize all streams
    hipDeviceSynchronize();

    if (with_grad) {

      // --------------------- Backward ----------------------------
      // initialization
      intialArrayGPU<<<blocks, threads>>>(d_vz_adj, nz, nx, 0.0);
      intialArrayGPU<<<blocks, threads>>>(d_vx_adj, nz, nx, 0.0);
      intialArrayGPU<<<blocks, threads>>>(d_szz_adj, nz, nx, 0.0);
      intialArrayGPU<<<blocks, threads>>>(d_sxx_adj, nz, nx, 0.0);
      intialArrayGPU<<<blocks, threads>>>(d_sxz_adj, nz, nx, 0.0);
      intialArrayGPU<<<blocks, threads>>>(d_mem_dvz_dz, nz, nx, 0.0);
      intialArrayGPU<<<blocks, threads>>>(d_mem_dvz_dx, nz, nx, 0.0);
      intialArrayGPU<<<blocks, threads>>>(d_mem_dvx_dz, nz, nx, 0.0);
      intialArrayGPU<<<blocks, threads>>>(d_mem_dvx_dx, nz, nx, 0.0);
      intialArrayGPU<<<blocks, threads>>>(d_mem_dszz_dz, nz, nx, 0.0);
      intialArrayGPU<<<blocks, threads>>>(d_mem_dsxz_dx, nz, nx, 0.0);
      intialArrayGPU<<<blocks, threads>>>(d_mem_dsxz_dz, nz, nx, 0.0);
      intialArrayGPU<<<blocks, threads>>>(d_mem_dsxx_dx, nz, nx, 0.0);
      intialArrayGPU<<<blocks, threads>>>(model.d_StfGrad, nt, 1, 0.0);
      initialArray(model.h_StfGrad, nt, 0.0);

      if (is_acoustic) {
        intialArrayGPU<<<blocks, threads>>>(d_szz_p1, nz, nx, 0.0);
      }

      if (is_acoustic) {

        // std::cout << "Backward modeling: acoustic" << std::endl;

        for (int it = nt - 2; it >= 0; it--) {
          // source time function kernels
          source_grad<<<1, 1>>>(d_szz_adj, d_sxx_adj, nz, model.d_StfGrad, it,
                                dt, survey.vec_z_src.at(iShot),
                                survey.vec_x_src.at(iShot));

          // save p to szz_plus_one
          assignArrayGPU<<<blocks, threads>>>(d_szz, d_szz_p1, nz, nx);

          // value at T-1
          acoustic_backward_velocity<<<blocks, threads>>>(
              d_vz, d_vx, d_szz, model.d_ave_Byc_a, model.d_ave_Byc_b, nz, nx,
              dt, dz, dx, npml, npad);

          // boundary values
          boundaries.field_to_bnd(d_szz, d_sxz, d_sxx, d_vz, d_vx, it, false);

          // subtract source
          sub_source<<<1, threads>>>(d_szz, d_sxx, survey.vec_source.at(iShot)[it],
                               nz, survey.vec_z_src.at(iShot),
                               survey.vec_x_src.at(iShot), dt, d_gauss_amp);

          // subtract source
          sub_source<<<1, threads>>>(d_szz_p1, d_sxx, survey.vec_source.at(iShot)[it],
                               nz, survey.vec_z_src.at(iShot),
                               survey.vec_x_src.at(iShot), dt, d_gauss_amp);

          acoustic_backward_pressure<<<blocks, threads>>>(
              d_vz, d_vx, d_szz, model.d_Lambda, nz, nx, dt, dz, dx, npml,
              npad);

          boundaries.field_to_bnd(d_szz, d_sxz, d_sxx, d_vz, d_vx, it, true);
          // value at T-2

          // adjoint computation
          acoustic_adjoint_velocity<<<blocks, threads>>>(
              d_vz_adj, d_vx_adj, d_szz_adj, d_mem_dvz_dz, d_mem_dvx_dx,
              d_mem_dszz_dz, d_mem_dsxx_dx, model.d_Lambda, model.d_ave_Byc_a,
              model.d_ave_Byc_b, cpml.d_K_z_half, cpml.d_a_z_half,
              cpml.d_b_z_half, cpml.d_K_x_half, cpml.d_a_x_half,
              cpml.d_b_x_half, cpml.d_K_z, cpml.d_a_z, cpml.d_b_z, cpml.d_K_x,
              cpml.d_a_x, cpml.d_b_x, nz, nx, dt, dz, dx, npml, npad);

          // inject geophone residuals
          inject_geo<<<(nrec + 31) / 32, 32>>>(
              d_szz_adj, d_sxx_adj, d_vx_adj, d_vz_adj, nz, d_res_pr, d_res_vx,
              d_res_vz, it, dt, nt, nrec, survey.d_vec_z_rec.at(iShot), 
              survey.d_vec_x_rec.at(iShot), weight_pr, weight_vx, weight_vz);
          
          // inject das residuals
          inject_das<<<(ndas + 31) / 32, 32>>>(d_vx_adj, d_vz_adj, nz, d_res_et, 
              it, dt, nt, ndas, survey.d_vec_z_das.at(iShot), 
              survey.d_vec_x_das.at(iShot), weight_et, 
              survey.d_vec_das_wt_x.at(iShot), 
              survey.d_vec_das_wt_z.at(iShot), survey.gl);

          acoustic_adjoint_pressure<<<blocks, threads>>>(
              d_vz_adj, d_vx_adj, d_szz_adj, d_mem_dvz_dz, d_mem_dvx_dx,
              d_mem_dszz_dz, d_mem_dsxx_dx, model.d_Lambda, model.d_ave_Byc_a,
              model.d_ave_Byc_b, cpml.d_K_z_half, cpml.d_a_z_half,
              cpml.d_b_z_half, cpml.d_K_x_half, cpml.d_a_x_half,
              cpml.d_b_x_half, cpml.d_K_z, cpml.d_a_z, cpml.d_b_z, cpml.d_K_x,
              cpml.d_a_x, cpml.d_b_x, nz, nx, dt, dz, dx, npml, npad);

          // value at T-1

          // imaging condition
          image_vel_time<<<blocks, threads>>>(
              d_szz, d_szz_p1, d_szz_adj, nz, nx, dt, dz, dx, npml, npad,
              model.d_Vp, model.d_Lambda, model.d_VpGrad);
        }
      } else {
        // std::cout << "Backward modeling: elastic" << std::endl;

        for (int it = nt - 2; it >= 0; it--) {
          // source time function kernels
          source_grad<<<1, 1>>>(d_szz_adj, d_sxx_adj, nz, model.d_StfGrad, it,
                                dt, survey.vec_z_src.at(iShot),
                                survey.vec_x_src.at(iShot));

          // update velocity
          elastic_backward_velocity<<<blocks, threads>>>(
              d_vz, d_vx, d_szz, d_sxx, d_sxz, model.d_ave_Byc_a,
              model.d_ave_Byc_b, nz, nx, dt, dz, dx, npml, npad, d_vz_adj,
              d_vx_adj, model.d_DenGrad);

          // inject boundary wavefields for reconstruction
          boundaries.field_to_bnd(d_szz, d_sxz, d_sxx, d_vz, d_vx, it, false);

          // subtract the source
          sub_source<<<1, threads>>>(
              d_szz, d_sxx, survey.vec_source.at(iShot)[it], nz,
              survey.vec_z_src.at(iShot), survey.vec_x_src.at(iShot), dt,
              d_gauss_amp);

          // update stress
          elastic_backward_stress<<<blocks, threads>>>(
              d_vz, d_vx, d_szz, d_sxx, d_sxz, model.d_Lambda, model.d_Mu,
              model.d_ave_Mu, nz, nx, dt, dz, dx, npml, npad, d_szz_adj,
              d_sxx_adj, d_sxz_adj, model.d_LambdaGrad, model.d_MuGrad);

          // inject boundary wavefields for reconstruction
          boundaries.field_to_bnd(d_szz, d_sxz, d_sxx, d_vz, d_vx, it, true);

          // update velocity of the adjoint wavefield
          elastic_adjoint_velocity<<<blocks, threads>>>(
              d_vz_adj, d_vx_adj, d_szz_adj, d_sxx_adj, d_sxz_adj,
              d_mem_dszz_dz, d_mem_dsxz_dx, d_mem_dsxz_dz, d_mem_dsxx_dx,
              d_mem_dvz_dz, d_mem_dvz_dx, d_mem_dvx_dz, d_mem_dvx_dx,
              model.d_Lambda, model.d_Mu, model.d_ave_Mu, model.d_Den,
              model.d_ave_Byc_a, model.d_ave_Byc_b, cpml.d_K_z_half,
              cpml.d_a_z_half, cpml.d_b_z_half, cpml.d_K_x_half,
              cpml.d_a_x_half, cpml.d_b_x_half, cpml.d_K_z, cpml.d_a_z,
              cpml.d_b_z, cpml.d_K_x, cpml.d_a_x, cpml.d_b_x, nz, nx, dt, dz,
              dx, npml, npad);

          // inject geophone residuals
          inject_geo<<<(nrec + 31) / 32, 32>>>(
            d_szz_adj, d_sxx_adj, d_vx_adj, d_vz_adj, nz, d_res_pr, d_res_vx,
            d_res_vz, it, dt, nt, nrec, survey.d_vec_z_rec.at(iShot), 
            survey.d_vec_x_rec.at(iShot), weight_pr, weight_vx, weight_vz);
        
          // inject das residuals
          inject_das<<<(ndas + 31) / 32, 32>>>(d_vx_adj, d_vz_adj, nz, d_res_et, 
              it, dt, nt, ndas, survey.d_vec_z_das.at(iShot), 
              survey.d_vec_x_das.at(iShot), weight_et, 
              survey.d_vec_das_wt_x.at(iShot),  
              survey.d_vec_das_wt_z.at(iShot), survey.gl);
          
          // update velocity of the adjoint wavefield
          elastic_adjoint_stress<<<blocks, threads>>>(
              d_vz_adj, d_vx_adj, d_szz_adj, d_sxx_adj, d_sxz_adj,
              d_mem_dszz_dz, d_mem_dsxz_dx, d_mem_dsxz_dz, d_mem_dsxx_dx,
              d_mem_dvz_dz, d_mem_dvz_dx, d_mem_dvx_dz, d_mem_dvx_dx,
              model.d_Lambda, model.d_Mu, model.d_ave_Mu, model.d_Den,
              model.d_ave_Byc_a, model.d_ave_Byc_b, cpml.d_K_z_half,
              cpml.d_a_z_half, cpml.d_b_z_half, cpml.d_K_x_half,
              cpml.d_a_x_half, cpml.d_b_x_half, cpml.d_K_z, cpml.d_a_z,
              cpml.d_b_z, cpml.d_K_x, cpml.d_a_x, cpml.d_b_x, nz, nx, dt, dz,
              dx, npml, npad);
        

          // Save wavefield for debug
          //if (iShot == 0 && it % 200 == 0) {
          //  CHECK(hipMemcpy(h_snap, d_vz_adj, nz * nx * sizeof(float), hipMemcpyDeviceToHost));
          //  fileBinWrite(h_snap, nz * nx, "SnapGPU_" + std::to_string(it) + ".bin");
          //}

        } // the end of backward time loop
      }
      
      // transfer source gradient to cpu
      CHECK(hipMemcpy(model.h_StfGrad, model.d_StfGrad, nt * sizeof(float),
                       hipMemcpyDeviceToHost));

      for (int it = 0; it < nt; it++) {
        grad_stf[iShot * nt + it] = model.h_StfGrad[it];
      }

    } // end bracket of if (with_grad)

    // free memory on GPU
    CHECK(hipFree(d_syn_pr));
    CHECK(hipFree(d_syn_vx));
    CHECK(hipFree(d_syn_vz));
    CHECK(hipFree(d_syn_et));

    // free memory on CPU
    if (with_grad) {
      CHECK(hipFree(d_obs_pr));
      CHECK(hipFree(d_obs_vx));
      CHECK(hipFree(d_obs_vz));
      CHECK(hipFree(d_obs_et));
      CHECK(hipFree(d_res_pr));
      CHECK(hipFree(d_res_vx));
      CHECK(hipFree(d_res_vz));
      CHECK(hipFree(d_res_et));
    }

  } // the end of shot loop

  if (with_grad) {
    if (is_acoustic) {
      // transfer gradients to cpu
      CHECK(hipMemcpy(model.h_VpGrad, model.d_VpGrad, nz * nx * sizeof(float),
                       hipMemcpyDeviceToHost));
      for (int i = 0; i < nz; i++) {
        for (int j = 0; j < nx; j++) {
          grad_Vp[i * nx + j] = model.h_VpGrad[j * nz + i];
        }
      }
    } else {
      // std::cout << "Transfer gradients to CPU for elastic gradient" << std::endl;

      // transfer gradients to cpu
      CHECK(hipMemcpy(model.h_LambdaGrad, model.d_LambdaGrad,
                       nz * nx * sizeof(float), hipMemcpyDeviceToHost));
      CHECK(hipMemcpy(model.h_MuGrad, model.d_MuGrad, nz * nx * sizeof(float),
                       hipMemcpyDeviceToHost));
      CHECK(hipMemcpy(model.h_DenGrad, model.d_DenGrad,
                       nz * nx * sizeof(float), hipMemcpyDeviceToHost));
      for (int i = 0; i < nz; i++) {
        for (int j = 0; j < nx; j++) {
          grad_Lambda[i * nx + j] = model.h_LambdaGrad[j * nz + i];
          grad_Mu[i * nx + j] = model.h_MuGrad[j * nz + i];
          grad_Den[i * nx + j] = model.h_DenGrad[j * nz + i];
        }
      }
    }

    if (para.if_save_scratch()) {
      for (int iShot = 0; iShot < group_size; iShot++) {
        // write the residual data to disk (processed)
        fileBinWrite(survey.vec_res_pr.at(iShot),
                     nt * survey.vec_nrec.at(iShot),
                     para.scratch_dir_name() + "/res_proc_shot" +
                         std::to_string(shot_ids[iShot]) + "_pr.bin");
        fileBinWrite(survey.vec_res_vx.at(iShot),
                     nt * survey.vec_nrec.at(iShot),
                     para.scratch_dir_name() + "/res_proc_shot" +
                         std::to_string(shot_ids[iShot]) + "_vx.bin");
        fileBinWrite(survey.vec_res_vz.at(iShot),
                     nt * survey.vec_nrec.at(iShot),
                     para.scratch_dir_name() + "/res_proc_shot" +
                         std::to_string(shot_ids[iShot]) + "_vz.bin");
        fileBinWrite(survey.vec_res_et.at(iShot),
                     nt * survey.vec_ndas.at(iShot),
                     para.scratch_dir_name() + "/res_proc_shot" +
                         std::to_string(shot_ids[iShot]) + "_et.bin");
        // write the synthetic data to disk (processed)
        fileBinWrite(survey.vec_syn_pr.at(iShot),
                     nt * survey.vec_nrec.at(iShot),
                     para.scratch_dir_name() + "/syn_proc_shot" +
                         std::to_string(shot_ids[iShot]) + "_pr.bin");
        fileBinWrite(survey.vec_syn_vx.at(iShot),
                     nt * survey.vec_nrec.at(iShot),
                     para.scratch_dir_name() + "/syn_proc_shot" +
                         std::to_string(shot_ids[iShot]) + "_vx.bin");
        fileBinWrite(survey.vec_syn_vz.at(iShot),
                     nt * survey.vec_nrec.at(iShot),
                     para.scratch_dir_name() + "/syn_proc_shot" +
                         std::to_string(shot_ids[iShot]) + "_vz.bin");
        fileBinWrite(survey.vec_syn_et.at(iShot),
                     nt * survey.vec_ndas.at(iShot),
                     para.scratch_dir_name() + "/syn_proc_shot" +
                         std::to_string(shot_ids[iShot]) + "_et.bin");
        // write the observed data to disk (processed)
        fileBinWrite(survey.vec_obs_pr.at(iShot),
                     nt * survey.vec_nrec.at(iShot),
                     para.scratch_dir_name() + "/obs_proc_shot" +
                         std::to_string(shot_ids[iShot]) + "_pr.bin");
        fileBinWrite(survey.vec_obs_vx.at(iShot),
                     nt * survey.vec_nrec.at(iShot),
                     para.scratch_dir_name() + "/obs_proc_shot" +
                         std::to_string(shot_ids[iShot]) + "_vx.bin");
        fileBinWrite(survey.vec_obs_vz.at(iShot),
                     nt * survey.vec_nrec.at(iShot),
                     para.scratch_dir_name() + "/obs_proc_shot" +
                         std::to_string(shot_ids[iShot]) + "_vz.bin");
        fileBinWrite(survey.vec_obs_et.at(iShot),
                     nt * survey.vec_ndas.at(iShot),
                     para.scratch_dir_name() + "/obs_proc_shot" +
                         std::to_string(shot_ids[iShot]) + "_et.bin");

        if (para.if_src_update()) {
          fileBinWrite(survey.vec_source.at(iShot), nt,
                       para.scratch_dir_name() + "/src_updated" +
                           std::to_string(shot_ids[iShot]) + ".bin");
        }
      }
    }
  }

  // output residual
  h_obj = 0.5 * h_obj;
  *misfit = h_obj;

  // free memory
  free(h_obj_pr);
  free(h_obj_vx);
  free(h_obj_vz);
  free(h_obj_et);
  free(fLambda);
  free(fMu);
  free(fDen);
  //free(h_snap);

  // destroy the streams
  for (int iShot = 0; iShot < group_size; iShot++) {
    CHECK(hipStreamDestroy(streams[iShot]));
  }

  hipFree(d_vz);
  hipFree(d_vx);
  hipFree(d_szz);
  hipFree(d_sxx);
  hipFree(d_sxz);
  hipFree(d_vz_adj);
  hipFree(d_vx_adj);
  hipFree(d_szz_adj);
  hipFree(d_sxx_adj);
  hipFree(d_sxz_adj);
  hipFree(d_mem_dvz_dz);
  hipFree(d_mem_dvz_dx);
  hipFree(d_mem_dvx_dz);
  hipFree(d_mem_dvx_dx);
  hipFree(d_mem_dszz_dz);
  hipFree(d_mem_dsxx_dx);
  hipFree(d_mem_dsxz_dz);
  hipFree(d_mem_dsxz_dx);
  hipFree(d_obj_pr);
  hipFree(d_obj_vx);
  hipFree(d_obj_vz);
  hipFree(d_obj_et);
  hipFree(d_gauss_amp);
}
