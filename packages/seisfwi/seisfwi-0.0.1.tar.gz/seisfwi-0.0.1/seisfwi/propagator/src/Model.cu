#include "Model.h"
#include "Parameter.h"
#include "utils.h"
#include <iostream>
#include <string>

// model default constructor
Model::Model() {
  std::cout << "ERROR: You need to supply parameters to initialize models!"
            << std::endl;
  exit(1);
}

// model constructor from parameter file
Model::Model(const Parameter &para, const float *Lambda_, const float *Mu_,
             const float *Den_) {

  nz_ = para.nz();
  nx_ = para.nx();

  dim3 threads(32, 16);
  dim3 blocks((nz_ + 32 - 1) / 32, (nx_ + 16 - 1) / 16);

  h_Lambda = (float *)malloc(nz_ * nx_ * sizeof(float));
  h_Mu = (float *)malloc(nz_ * nx_ * sizeof(float));
  h_Den = (float *)malloc(nz_ * nx_ * sizeof(float));
  h_Vp = (float *)malloc(nz_ * nx_ * sizeof(float));
  h_Vs = (float *)malloc(nz_ * nx_ * sizeof(float));
  h_VpGrad = (float *)malloc(nz_ * nx_ * sizeof(float));
  h_LambdaGrad = (float *)malloc(nz_ * nx_ * sizeof(float));
  h_MuGrad = (float *)malloc(nz_ * nx_ * sizeof(float));
  h_DenGrad = (float *)malloc(nz_ * nx_ * sizeof(float));
  h_StfGrad = (float *)malloc(para.nt() * sizeof(float));


  for (int i = 0; i < nz_ * nx_; i++) {
    if (Lambda_[i] < 0.0) {
      printf("Lambda is negative!");

      // exit program
      exit(1);
    }
    h_Lambda[i] = Lambda_[i];
    h_Mu[i] = Mu_[i];
    h_Den[i] = Den_[i];
  }

  initialArray(h_Vp, nz_ * nx_, 0.0);
  initialArray(h_Vs, nz_ * nx_, 0.0);
  initialArray(h_VpGrad, nz_ * nx_, 0.0);
  initialArray(h_LambdaGrad, nz_ * nx_, 0.0);
  initialArray(h_MuGrad, nz_ * nx_, 0.0);
  initialArray(h_DenGrad, nz_ * nx_, 0.0);
  initialArray(h_StfGrad, para.nt(), 0.0);

  CHECK(hipMalloc((void **)&d_Lambda, nz_ * nx_ * sizeof(float)));
  CHECK(hipMalloc((void **)&d_Mu, nz_ * nx_ * sizeof(float)));
  CHECK(hipMalloc((void **)&d_Den, nz_ * nx_ * sizeof(float)));
  CHECK(hipMalloc((void **)&d_Vp, nz_ * nx_ * sizeof(float)));
  CHECK(hipMalloc((void **)&d_Vs, nz_ * nx_ * sizeof(float)));
  CHECK(hipMalloc((void **)&d_ave_Mu, nz_ * nx_ * sizeof(float)));
  CHECK(hipMalloc((void **)&d_ave_Byc_a, nz_ * nx_ * sizeof(float)));
  CHECK(hipMalloc((void **)&d_ave_Byc_b, nz_ * nx_ * sizeof(float)));
  CHECK(hipMalloc((void **)&d_VpGrad, nz_ * nx_ * sizeof(float)));
  CHECK(hipMalloc((void **)&d_LambdaGrad, nz_ * nx_ * sizeof(float)));
  CHECK(hipMalloc((void **)&d_MuGrad, nz_ * nx_ * sizeof(float)));
  CHECK(hipMalloc((void **)&d_DenGrad, nz_ * nx_ * sizeof(float)));
  CHECK(hipMalloc((void **)&d_StfGrad, para.nt() * sizeof(float)));
  intialArrayGPU<<<blocks, threads>>>(d_ave_Mu, nz_, nx_, 0.0);
  intialArrayGPU<<<blocks, threads>>>(d_VpGrad, nz_, nx_, 0.0);
  intialArrayGPU<<<blocks, threads>>>(d_LambdaGrad, nz_, nx_, 0.0);
  intialArrayGPU<<<blocks, threads>>>(d_MuGrad, nz_, nx_, 0.0);
  intialArrayGPU<<<blocks, threads>>>(d_DenGrad, nz_, nx_, 0.0);
  intialArrayGPU<<<blocks, threads>>>(d_ave_Byc_a, nz_, nx_, 1.0 / 1000.0);
  intialArrayGPU<<<blocks, threads>>>(d_ave_Byc_b, nz_, nx_, 1.0 / 1000.0);
  intialArrayGPU<<<blocks, threads>>>(d_StfGrad, para.nt(), 1, 0.0);

  CHECK(hipMemcpy(d_Lambda, h_Lambda, nz_ * nx_ * sizeof(float),
                   hipMemcpyHostToDevice));
  CHECK(hipMemcpy(d_Mu, h_Mu, nz_ * nx_ * sizeof(float),
                   hipMemcpyHostToDevice));
  CHECK(hipMemcpy(d_Den, h_Den, nz_ * nx_ * sizeof(float),
                   hipMemcpyHostToDevice));

  velInit<<<blocks, threads>>>(d_Lambda, d_Mu, d_Den, d_Vp, d_Vs, nz_, nx_);
  aveMuInit<<<blocks, threads>>>(d_Mu, d_ave_Mu, nz_, nx_);
  aveBycInit<<<blocks, threads>>>(d_Den, d_ave_Byc_a, d_ave_Byc_b, nz_, nx_);

  CHECK(hipMemcpy(h_Vp, d_Vp, nz_ * nx_ * sizeof(float),
                   hipMemcpyDeviceToHost));
  CHECK(hipMemcpy(h_Vs, d_Vs, nz_ * nx_ * sizeof(float),
                   hipMemcpyDeviceToHost));

}

Model::~Model() {
  free(h_Vp);
  free(h_Vs);
  free(h_Den);
  free(h_Lambda);
  free(h_Mu);
  free(h_VpGrad);
  CHECK(hipFree(d_Vp));
  CHECK(hipFree(d_Vs));
  CHECK(hipFree(d_Den));
  CHECK(hipFree(d_Lambda));
  CHECK(hipFree(d_Mu));
  CHECK(hipFree(d_ave_Mu));
  CHECK(hipFree(d_ave_Byc_a));
  CHECK(hipFree(d_ave_Byc_b));
  CHECK(hipFree(d_VpGrad));
  CHECK(hipFree(d_LambdaGrad));
  CHECK(hipFree(d_MuGrad));
  CHECK(hipFree(d_DenGrad));
  CHECK(hipFree(d_StfGrad));
}