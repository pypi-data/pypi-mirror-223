#include "Cpml.h"
#include "Model.h"
#include "Parameter.h"
#include "utils.h"

Cpml::Cpml(Parameter &para, Model &model) {

  int nz = model.nz();
  int nx = model.nx();
  int npml = para.npml();
  int npad = para.npad();
  float f0 = para.f0();
  float dt = para.dt();
  float dz = para.dz();
  float dx = para.dx();

  float CpAve = compCpAve(model.h_Vp, nz * nx);

  // for padding
  K_z = (float *)malloc((nz - npad) * sizeof(float));
  a_z = (float *)malloc((nz - npad) * sizeof(float));
  b_z = (float *)malloc((nz - npad) * sizeof(float));
  K_z_half = (float *)malloc((nz - npad) * sizeof(float));
  a_z_half = (float *)malloc((nz - npad) * sizeof(float));
  b_z_half = (float *)malloc((nz - npad) * sizeof(float));

  K_x = (float *)malloc(nx * sizeof(float));
  a_x = (float *)malloc(nx * sizeof(float));
  b_x = (float *)malloc(nx * sizeof(float));
  K_x_half = (float *)malloc(nx * sizeof(float));
  a_x_half = (float *)malloc(nx * sizeof(float));
  b_x_half = (float *)malloc(nx * sizeof(float));

  cpmlInit(K_z, a_z, b_z, K_z_half, a_z_half, b_z_half, nz - npad, npml, dz, f0,
           dt, CpAve);

  cpmlInit(K_x, a_x, b_x, K_x_half, a_x_half, b_x_half, nx, npml, dx, f0, dt,
           CpAve);

  // for padding
  CHECK(hipMalloc((void **)&d_K_z, (nz - npad) * sizeof(float)));
  CHECK(hipMalloc((void **)&d_a_z, (nz - npad) * sizeof(float)));
  CHECK(hipMalloc((void **)&d_b_z, (nz - npad) * sizeof(float)));
  CHECK(hipMalloc((void **)&d_K_z_half, (nz - npad) * sizeof(float)));
  CHECK(hipMalloc((void **)&d_a_z_half, (nz - npad) * sizeof(float)));
  CHECK(hipMalloc((void **)&d_b_z_half, (nz - npad) * sizeof(float)));

  CHECK(hipMalloc((void **)&d_K_x, nx * sizeof(float)));
  CHECK(hipMalloc((void **)&d_a_x, nx * sizeof(float)));
  CHECK(hipMalloc((void **)&d_b_x, nx * sizeof(float)));
  CHECK(hipMalloc((void **)&d_K_x_half, nx * sizeof(float)));
  CHECK(hipMalloc((void **)&d_a_x_half, nx * sizeof(float)));
  CHECK(hipMalloc((void **)&d_b_x_half, nx * sizeof(float)));

  // for padding
  CHECK(hipMemcpy(d_K_z, K_z, (nz - npad) * sizeof(float),
                   hipMemcpyHostToDevice));
  CHECK(hipMemcpy(d_a_z, a_z, (nz - npad) * sizeof(float),
                   hipMemcpyHostToDevice));
  CHECK(hipMemcpy(d_b_z, b_z, (nz - npad) * sizeof(float),
                   hipMemcpyHostToDevice));
  CHECK(hipMemcpy(d_K_z_half, K_z_half, (nz - npad) * sizeof(float),
                   hipMemcpyHostToDevice));
  CHECK(hipMemcpy(d_a_z_half, a_z_half, (nz - npad) * sizeof(float),
                   hipMemcpyHostToDevice));
  CHECK(hipMemcpy(d_b_z_half, b_z_half, (nz - npad) * sizeof(float),
                   hipMemcpyHostToDevice));

  CHECK(hipMemcpy(d_K_x, K_x, nx * sizeof(float), hipMemcpyHostToDevice));
  CHECK(hipMemcpy(d_a_x, a_x, nx * sizeof(float), hipMemcpyHostToDevice));
  CHECK(hipMemcpy(d_b_x, b_x, nx * sizeof(float), hipMemcpyHostToDevice));
  CHECK(hipMemcpy(d_K_x_half, K_x_half, nx * sizeof(float),
                   hipMemcpyHostToDevice));
  CHECK(hipMemcpy(d_a_x_half, a_x_half, nx * sizeof(float),
                   hipMemcpyHostToDevice));
  CHECK(hipMemcpy(d_b_x_half, b_x_half, nx * sizeof(float),
                   hipMemcpyHostToDevice));
}

Cpml::~Cpml() {
  free(K_z);
  free(a_z);
  free(b_z);
  free(K_z_half);
  free(a_z_half);
  free(b_z_half);
  free(K_x);
  free(a_x);
  free(b_x);
  free(K_x_half);
  free(a_x_half);
  free(b_x_half);

  CHECK(hipFree(d_K_z));
  CHECK(hipFree(d_a_z));
  CHECK(hipFree(d_b_z));
  CHECK(hipFree(d_K_z_half));
  CHECK(hipFree(d_a_z_half));
  CHECK(hipFree(d_b_z_half));
  CHECK(hipFree(d_K_x));
  CHECK(hipFree(d_a_x));
  CHECK(hipFree(d_b_x));
  CHECK(hipFree(d_K_x_half));
  CHECK(hipFree(d_a_x_half));
  CHECK(hipFree(d_b_x_half));
}