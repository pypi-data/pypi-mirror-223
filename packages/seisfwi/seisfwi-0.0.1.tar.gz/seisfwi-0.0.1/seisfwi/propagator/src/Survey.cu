// Dongzhuo Li 05/13/2018
// Modified by Dongzhuo Li 06/25/2018
#include "Survey.h"
#include "utils.h"
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <string>

#define stf(iShot, it) stf[(iShot) * (nt) + (it)] // row-major

using namespace std;
using namespace rapidjson;

Survey::Survey() {
  cout << "ERROR: You need to input parameter!" << endl;
  exit(1);
}

Survey::Survey(Parameter &para, const bool with_adj, const bool with_residual,
               const float *stf, int group_size, const int *shot_ids) {

  // default values
  string line;
  ifstream survery_file;

  // open survey file and read in the survey info
  survery_file.open(para.survey_fname());

  if (!survery_file.is_open()) {
    cout << "Error opening survey file" << endl;
    exit(1);
  }

  getline(survery_file, line);
  survery_file.close();

  Document json_survery;
  json_survery.Parse<0>(line.c_str());
  assert(json_survery.IsObject());

  // surver mode
  with_adj_ = with_adj;
  with_residual_ = with_residual;
  if_win_ = para.if_win();

  int nrec = 0;
  int ndas = 0;
  int z_src = 10;
  int x_src = 10;
  int nt = para.nt();
  int npml = para.npml();
  float dt = para.dt();

  int *h_z_rec = nullptr;
  int *h_x_rec = nullptr;
  int *h_z_das = nullptr;
  int *h_x_das = nullptr;
  
  float *h_win_start = nullptr; // for selected window
  float *h_win_end = nullptr;   // for selected window
  float *h_x_weight = nullptr;   // for das weighting
  float *h_z_weight = nullptr;   // for das weighting
  float *h_source = nullptr;

  // syn data: pr, vx, vz, et components
  float *h_syn_pr = nullptr;
  float *h_syn_vx = nullptr;
  float *h_syn_vz = nullptr;
  float *h_syn_et = nullptr;
  // obs data: pr, vx, vz, et components
  float *h_obs_pr = nullptr;
  float *h_obs_vx = nullptr;
  float *h_obs_vz = nullptr;
  float *h_obs_et = nullptr;
  // res data: pr, vx, vz, et components
  float *h_res_pr = nullptr;
  float *h_res_vx = nullptr;
  float *h_res_vz = nullptr;
  float *h_res_et = nullptr;
  // adj data: pr components since explosive source
  float *h_adj_pr = nullptr;

  // for shot number small than 99999
  char thisShot[10];

  // device pointers
  int *d_z_rec, *d_x_rec;
  int *d_z_das, *d_x_das;
  float *d_source;
  float *d_win_start, *d_win_end;
  float *d_x_weight, *d_z_weight;


  // get the source number
  assert(json_survery.HasMember("nShots"));
  assert(json_survery["nShots"].IsInt());
  nShots = json_survery["nShots"].GetInt();

  // get the guage length
  assert(json_survery.HasMember("gauge_length"));
  assert(json_survery["gauge_length"].IsFloat());
  gl = json_survery["gauge_length"].GetFloat();

  CHECK(hipMalloc((void **)&d_coef, (nt + 1) * sizeof(hipFloatComplex)));

  for (int i = 0; i < group_size; i++) {

    // get the source positions: z_src, x_src
    strcpy(thisShot, ("shot" + to_string(shot_ids[i])).c_str());

    assert(json_survery[thisShot].HasMember("z_src"));
    assert(json_survery[thisShot]["z_src"].IsInt());
    z_src = json_survery[thisShot]["z_src"].GetInt() + npml;
    vec_z_src.push_back(z_src);

    assert(json_survery[thisShot].HasMember("x_src"));
    assert(json_survery[thisShot]["x_src"].IsInt());
    x_src = json_survery[thisShot]["x_src"].GetInt() + npml;
    vec_x_src.push_back(x_src);

    // get the number of rec for each shot
    assert(json_survery[thisShot].HasMember("nrec"));
    assert(json_survery[thisShot]["nrec"].IsInt());
    nrec = json_survery[thisShot]["nrec"].GetInt();
    vec_nrec.push_back(nrec);
    h_z_rec = new int[nrec];
    h_x_rec = new int[nrec];

    // read in the receiver positions for this shot: z_rec, x_rec
    assert(json_survery[thisShot].HasMember("z_rec"));
    assert(json_survery[thisShot]["z_rec"].IsArray());
    const Value &js_z_rec = json_survery[thisShot]["z_rec"];
    for (SizeType ii = 0; ii < js_z_rec.Size(); ii++) {
      h_z_rec[ii] = js_z_rec[ii].GetInt() + npml;
    }

    assert(json_survery[thisShot].HasMember("x_rec"));
    assert(json_survery[thisShot]["x_rec"].IsArray());
    const Value &js_x_rec = json_survery[thisShot]["x_rec"];
    for (SizeType ii = 0; ii < js_x_rec.Size(); ii++) {
      h_x_rec[ii] = js_x_rec[ii].GetInt() + npml;
    }

    // get receiver z positions for each shot
    CHECK(hipMalloc((void **)&d_z_rec, nrec * sizeof(int)));
    CHECK(hipMemcpy(d_z_rec, h_z_rec, nrec * sizeof(int), hipMemcpyHostToDevice));
    d_vec_z_rec.push_back(d_z_rec);

    // get receiver x positions for each shot
    CHECK(hipMalloc((void **)&d_x_rec, nrec * sizeof(int)));
    CHECK(hipMemcpy(d_x_rec, h_x_rec, nrec * sizeof(int), hipMemcpyHostToDevice));
    d_vec_x_rec.push_back(d_x_rec);

  
    // get the number of DAS channel for each shot
    assert(json_survery[thisShot].HasMember("ndas"));
    assert(json_survery[thisShot]["ndas"].IsInt());
    ndas = json_survery[thisShot]["ndas"].GetInt();
    vec_ndas.push_back(ndas);
    h_z_das = new int[ndas];
    h_x_das = new int[ndas];
  
    // read in the DAS channel positions for this shot: z_das, x_das
    assert(json_survery[thisShot].HasMember("z_das"));
    assert(json_survery[thisShot]["z_das"].IsArray());
    const Value &js_z_das = json_survery[thisShot]["z_das"];
    for (SizeType ii = 0; ii < js_z_das.Size(); ii++) {
      h_z_das[ii] = js_z_das[ii].GetInt() + npml;
    }

    assert(json_survery[thisShot].HasMember("x_das"));
    assert(json_survery[thisShot]["x_das"].IsArray());
    const Value &js_x_das = json_survery[thisShot]["x_das"];
    for (SizeType ii = 0; ii < js_x_das.Size(); ii++) {
      h_x_das[ii] = js_x_das[ii].GetInt() + npml;
    }

    // get DAS z positions for each shot
    CHECK(hipMalloc((void **)&d_z_das, ndas * sizeof(int)));
    CHECK(hipMemcpy(d_z_das, h_z_das, ndas * sizeof(int), hipMemcpyHostToDevice));
    d_vec_z_das.push_back(d_z_das);

    // get DAS x positions for each shot
    CHECK(hipMalloc((void **)&d_x_das, ndas * sizeof(int)));
    CHECK(hipMemcpy(d_x_das, h_x_das, ndas * sizeof(int), hipMemcpyHostToDevice));
    d_vec_x_das.push_back(d_x_das);
    

    // get weights: the weight is for x component
    h_x_weight = new float[ndas];
    assert(json_survery[thisShot].HasMember("das_wt_x"));
    assert(json_survery[thisShot]["das_wt_x"].IsArray());
    const Value &js_weights = json_survery[thisShot]["das_wt_x"];
    for (SizeType ii = 0; ii < js_weights.Size(); ii++) {
      h_x_weight[ii] = js_weights[ii].GetDouble();
    }
    CHECK(hipMalloc((void **)&d_x_weight, ndas * sizeof(float)));
    CHECK(hipMemcpy(d_x_weight, h_x_weight, ndas * sizeof(float), hipMemcpyHostToDevice));
    d_vec_das_wt_x.push_back(d_x_weight);
    delete[] h_x_weight;

    // get weights: the weight is for z component
    h_z_weight = new float[ndas];
    assert(json_survery[thisShot].HasMember("das_wt_z"));
    assert(json_survery[thisShot]["das_wt_z"].IsArray());
    const Value &js_weights2 = json_survery[thisShot]["das_wt_z"];
    for (SizeType ii = 0; ii < js_weights2.Size(); ii++) {
      h_z_weight[ii] = js_weights2[ii].GetDouble();
    }
    CHECK(hipMalloc((void **)&d_z_weight, ndas * sizeof(float)));
    CHECK(hipMemcpy(d_z_weight, h_z_weight, ndas * sizeof(float), hipMemcpyHostToDevice));
    d_vec_das_wt_z.push_back(d_z_weight);
    delete[] h_z_weight;


    // get the source time function for each shot
    h_source = new float[nt];
    for (int it = 0; it < nt; it++) {
      h_source[it] = stf(shot_ids[i], it);
    }

    CHECK(hipMalloc((void **)&d_source, nt * sizeof(float)));
    CHECK(hipMemcpy(d_source, h_source, nt * sizeof(float), hipMemcpyHostToDevice));
    cuda_window<<<(nt + 31) / 32, 32>>>(nt, 1, dt, 0.001, d_source);

    // if (para.if_filter()) {
    //   bp_filter1d(nt, dt, 1, d_source, para.filter());
    // }

    CHECK(hipMemcpy(h_source, d_source, nt * sizeof(float), hipMemcpyDeviceToHost));
    vec_source.push_back(h_source);
    d_vec_source.push_back(d_source);

    // get the window for each shot
    if (if_win_) {
      h_win_start = new float[nrec];
      h_win_end = new float[nrec];

      // window start
      assert(json_survery[thisShot].HasMember("win_start"));
      assert(json_survery[thisShot]["win_start"].IsArray());
      const Value &js_win_start = json_survery[thisShot]["win_start"];
      for (SizeType ii = 0; ii < js_win_start.Size(); ii++) {
        h_win_start[ii] = js_win_start[ii].GetDouble();
      }

      // window end
      assert(json_survery[thisShot].HasMember("win_end"));
      assert(json_survery[thisShot]["win_end"].IsArray());
      const Value &js_win_end = json_survery[thisShot]["win_end"];
      for (SizeType ii = 0; ii < js_win_end.Size(); ii++) {
        h_win_end[ii] = js_win_end[ii].GetDouble();
      }

      // push to device
      CHECK(hipMalloc((void **)&d_win_start, nrec * sizeof(float)));
      CHECK(hipMemcpy(d_win_start, h_win_start, nrec * sizeof(float), hipMemcpyHostToDevice));
      d_vec_win_start.push_back(d_win_start);

      CHECK(hipMalloc((void **)&d_win_end, nrec * sizeof(float)));
      CHECK(hipMemcpy(d_win_end, h_win_end, nrec * sizeof(float), hipMemcpyHostToDevice));
      d_vec_win_end.push_back(d_win_end);

      delete[] h_win_start;
      delete[] h_win_end;
    }


    // Initialize the host side data cube for pr, vx, vz and et components
    hipHostAlloc((void **)&h_syn_pr, nt * nrec * sizeof(float), hipHostMallocDefault);
    initialArray(h_syn_pr, nt * nrec, 0.0);
    vec_syn_pr.push_back(h_syn_pr);

    hipHostAlloc((void **)&h_syn_vx, nt * nrec * sizeof(float), hipHostMallocDefault);
    initialArray(h_syn_vx, nt * nrec, 0.0);
    vec_syn_vx.push_back(h_syn_vx);

    hipHostAlloc((void **)&h_syn_vz, nt * nrec * sizeof(float), hipHostMallocDefault);
    initialArray(h_syn_vz, nt * nrec, 0.0);
    vec_syn_vz.push_back(h_syn_vz);

    hipHostAlloc((void **)&h_syn_et, nt * ndas * sizeof(float), hipHostMallocDefault);
    initialArray(h_syn_et, nt * ndas, 0.0);
    vec_syn_et.push_back(h_syn_et);

    if (with_residual_) {
      // initialize the host side observed data cube for pr, vx, vz and et
      // components
      hipHostAlloc((void **)&h_obs_pr, nt * nrec * sizeof(float), hipHostMallocDefault);
      initialArray(h_obs_pr, nt * nrec, 0.0);
      vec_obs_pr.push_back(h_obs_pr);

      hipHostAlloc((void **)&h_obs_vx, nt * nrec * sizeof(float), hipHostMallocDefault);
      initialArray(h_obs_vx, nt * nrec, 0.0);
      vec_obs_vx.push_back(h_obs_vx);

      hipHostAlloc((void **)&h_obs_vz, nt * nrec * sizeof(float), hipHostMallocDefault);
      initialArray(h_obs_vz, nt * nrec, 0.0);
      vec_obs_vz.push_back(h_obs_vz);

      hipHostAlloc((void **)&h_obs_et, nt * ndas * sizeof(float), hipHostMallocDefault);
      initialArray(h_obs_et, nt * ndas, 0.0);
      vec_obs_et.push_back(h_obs_et);

      // initialize the host side data residual data cube for pr, vx, vz and et
      // components
      hipHostAlloc((void **)&h_res_pr, nt * nrec * sizeof(float), hipHostMallocDefault);
      initialArray(h_res_pr, nt * nrec, 0.0);
      vec_res_pr.push_back(h_res_pr);

      hipHostAlloc((void **)&h_res_vx, nt * nrec * sizeof(float), hipHostMallocDefault);
      initialArray(h_res_vx, nt * nrec, 0.0);
      vec_res_vx.push_back(h_res_vx);

      hipHostAlloc((void **)&h_res_vz, nt * nrec * sizeof(float), hipHostMallocDefault);
      initialArray(h_res_vz, nt * nrec, 0.0);
      vec_res_vz.push_back(h_res_vz);

      hipHostAlloc((void **)&h_res_et, nt * ndas * sizeof(float), hipHostMallocDefault);
      initialArray(h_res_et, nt * ndas, 0.0);
      vec_res_et.push_back(h_res_et);
    }

    if (with_adj) {
      hipHostAlloc((void **)&h_adj_pr, nt * 1 * sizeof(float), hipHostMallocDefault);
      initialArray(h_adj_pr, nt * 1, 0.0);
      vec_adj_pr.push_back(h_adj_pr);
    }

  }
}

Survey::~Survey() {

  for (int i = 0; i < d_vec_x_rec.size(); i++) {
    CHECK(hipFree(d_vec_z_rec.at(i)));
    CHECK(hipFree(d_vec_x_rec.at(i)));
  }

  for (int i = 0; i < d_vec_x_das.size(); i++) {
    CHECK(hipFree(d_vec_z_das.at(i)));
    CHECK(hipFree(d_vec_x_das.at(i)));
  }

  for (int i = 0; i < vec_source.size(); i++) {
    delete[] vec_source.at(i);
    CHECK(hipFree(d_vec_source.at(i)));
  }

  // free forward record data
  for (int i = 0; i < vec_syn_pr.size(); i++) {
    CHECK(hipHostFree(vec_syn_pr.at(i)));
    CHECK(hipHostFree(vec_syn_vx.at(i)));
    CHECK(hipHostFree(vec_syn_vz.at(i)));
  }

  for (int i = 0; i < vec_syn_et.size(); i++) {
    CHECK(hipHostFree(vec_syn_et.at(i)));
  }

  // free residual record data
  if (with_residual_) {
    for (int i = 0; i < vec_obs_pr.size(); i++) {
      CHECK(hipHostFree(vec_obs_pr.at(i)));
      CHECK(hipHostFree(vec_obs_vx.at(i)));
      CHECK(hipHostFree(vec_obs_vz.at(i)));
      CHECK(hipHostFree(vec_res_pr.at(i)));
      CHECK(hipHostFree(vec_res_vx.at(i)));
      CHECK(hipHostFree(vec_res_vz.at(i)));
    }
    for (int i = 0; i < vec_obs_et.size(); i++) {
      CHECK(hipHostFree(vec_obs_et.at(i)));
      CHECK(hipHostFree(vec_res_et.at(i)));
    }
  }

  // free adjoint record data
  if (with_adj_) {
    for (int i = 0; i < vec_adj_pr.size(); i++) {
      CHECK(hipHostFree(vec_adj_pr.at(i)));
    }
  }

  // free windowing data
  if (if_win_) {
    for (int i = 0; i < d_vec_win_start.size(); i++) {
      CHECK(hipFree(d_vec_win_start.at(i)));
      CHECK(hipFree(d_vec_win_end.at(i)));
    }
  }

  // free source weight data
  for (int i = 0; i < d_vec_das_wt_x.size(); i++) {
    CHECK(hipFree(d_vec_das_wt_x.at(i)));
    CHECK(hipFree(d_vec_das_wt_z.at(i)));
  }

  CHECK(hipFree(d_coef));
}