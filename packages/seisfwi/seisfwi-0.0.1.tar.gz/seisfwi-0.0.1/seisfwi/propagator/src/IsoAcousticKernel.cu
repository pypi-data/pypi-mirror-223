#include "hip/hip_runtime.h"
#define d_vx(z, x) d_vx[(x) * (nz) + (z)]
#define d_vz(z, x) d_vz[(x) * (nz) + (z)]
#define d_szz(z, x) d_szz[(x) * (nz) + (z)] // Pressure
#define d_mem_dszz_dz(z, x) d_mem_dszz_dz[(x) * (nz) + (z)]
#define d_mem_dsxx_dx(z, x) d_mem_dsxx_dx[(x) * (nz) + (z)]
#define d_mem_dvz_dz(z, x) d_mem_dvz_dz[(x) * (nz) + (z)]
#define d_mem_dvx_dx(z, x) d_mem_dvx_dx[(x) * (nz) + (z)]
#define d_Lambda(z, x) d_Lambda[(x) * (nz) + (z)]
#define d_ave_Byc_a(z, x) d_ave_Byc_a[(x) * (nz) + (z)]
#define d_ave_Byc_b(z, x) d_ave_Byc_b[(x) * (nz) + (z)]
#define d_mat_dvz_dz(z, x) d_mat_dvz_dz[(x) * (nz) + (z)]
#define d_mat_dvx_dx(z, x) d_mat_dvx_dx[(x) * (nz) + (z)]
#define d_Vp(z, x) d_Vp[(x) * (nz) + (z)]
#define d_VpGrad(z, x) d_VpGrad[(x) * (nz) + (z)]
#define d_szz_plusone(z, x) d_szz_plusone[(x) * (nz) + (z)]
#define d_szz_adj(z, x) d_szz_adj[(x) * (nz) + (z)]

#include "utils.h"

// acoustic_forward_velocity
// acoustic_forward_stress
// acoustic_backward_stress
// acoustic_backward_velocity
// acoustic_forward_stress
// acoustic_forward_velocity

__global__ void
acoustic_forward_velocity(float *d_vz, float *d_vx, float *d_szz,
                          float *d_mem_dszz_dz, float *d_mem_dsxx_dx,
                          float *d_ave_Byc_a, float *d_ave_Byc_b, float *d_K_z,
                          float *d_a_z, float *d_b_z, float *d_K_x_half,
                          float *d_a_x_half, float *d_b_x_half, int nz, int nx,
                          float dt, float dz, float dx, int npml, int npad) {

  // global index
  int gidz = blockIdx.x * blockDim.x + threadIdx.x;
  int gidx = blockIdx.y * blockDim.y + threadIdx.y;

  float dszz_dz = 0.0;
  float dsxx_dx = 0.0;

  float c1 = 9.0 / 8.0;
  float c2 = 1.0 / 24.0;

  if (gidz >= 2 && gidz <= nz - npad - 3 && gidx >= 2 && gidx <= nx - 3) {

    // update vz
    dszz_dz = (c1 * (d_szz(gidz, gidx) - d_szz(gidz - 1, gidx)) -
               c2 * (d_szz(gidz + 1, gidx) - d_szz(gidz - 2, gidx))) /
              dz;

    if (gidz <= npml || (gidz >= nz - npml - npad - 1)) {
      d_mem_dszz_dz(gidz, gidx) =
          d_b_z[gidz] * d_mem_dszz_dz(gidz, gidx) + d_a_z[gidz] * dszz_dz;
    }

    d_vz(gidz, gidx) += (dszz_dz / d_K_z[gidz] + d_mem_dszz_dz(gidz, gidx)) *
                        d_ave_Byc_a(gidz, gidx) * dt;

    // update vx
    dsxx_dx = (c1 * (d_szz(gidz, gidx + 1) - d_szz(gidz, gidx)) -
               c2 * (d_szz(gidz, gidx + 2) - d_szz(gidz, gidx - 1))) /
              dx;

    if (gidx <= npml || gidx >= nx - npml - 1) {
      d_mem_dsxx_dx(gidz, gidx) = d_b_x_half[gidx] * d_mem_dsxx_dx(gidz, gidx) +
                                  d_a_x_half[gidx] * dsxx_dx;
    }

    d_vx(gidz, gidx) +=
        (dsxx_dx / d_K_x_half[gidx] + d_mem_dsxx_dx(gidz, gidx)) *
        d_ave_Byc_b(gidz, gidx) * dt;

  } else {
    return;
  }
}

__global__ void acoustic_forward_pressure(
    float *d_vz, float *d_vx, float *d_szz, float *d_mem_dvz_dz,
    float *d_mem_dvx_dx, float *d_Lambda, float *d_K_z_half, float *d_a_z_half,
    float *d_b_z_half, float *d_K_x, float *d_a_x, float *d_b_x, int nz, int nx,
    float dt, float dz, float dx, int npml, int npad) {

  int gidz = blockIdx.x * blockDim.x + threadIdx.x;
  int gidx = blockIdx.y * blockDim.y + threadIdx.y;

  float dvz_dz = 0.0;
  float dvx_dx = 0.0;

  float c1 = 9.0 / 8.0;
  float c2 = 1.0 / 24.0;

  if (gidz >= 2 && gidz <= nz - npad - 3 && gidx >= 2 && gidx <= nx - 3) {

    dvz_dz = (c1 * (d_vz(gidz + 1, gidx) - d_vz(gidz, gidx)) -
              c2 * (d_vz(gidz + 2, gidx) - d_vz(gidz - 1, gidx))) / dz;

    dvx_dx = (c1 * (d_vx(gidz, gidx) - d_vx(gidz, gidx - 1)) -
              c2 * (d_vx(gidz, gidx + 1) - d_vx(gidz, gidx - 2))) / dx;

    if (gidz <= npml || (gidz >= nz - npml - npad - 1)) {
      d_mem_dvz_dz(gidz, gidx) = d_b_z_half[gidz] * d_mem_dvz_dz(gidz, gidx) +
                                 d_a_z_half[gidz] * dvz_dz;
      dvz_dz = dvz_dz / d_K_z_half[gidz] + d_mem_dvz_dz(gidz, gidx);
    }
    if (gidx <= npml || gidx >= nx - npml - 1) {
      d_mem_dvx_dx(gidz, gidx) =
          d_b_x[gidx] * d_mem_dvx_dx(gidz, gidx) + d_a_x[gidx] * dvx_dx;
      dvx_dx = dvx_dx / d_K_x[gidx] + d_mem_dvx_dx(gidz, gidx);
    }

    d_szz(gidz, gidx) += d_Lambda(gidz, gidx) * (dvz_dz + dvx_dx) * dt;

  } else {
    return;
  }
}

__global__ void acoustic_backward_velocity(float *d_vz, float *d_vx,
                                           float *d_szz, float *d_ave_Byc_a,
                                           float *d_ave_Byc_b, int nz, int nx,
                                           float dt, float dz, float dx,
                                           int npml, int npad) {

  int gidz = blockIdx.x * blockDim.x + threadIdx.x;
  int gidx = blockIdx.y * blockDim.y + threadIdx.y;

  float dszz_dz = 0.0;
  float dsxx_dx = 0.0;

  float c1 = 9.0 / 8.0;
  float c2 = 1.0 / 24.0;

  if (gidz >= npml + 2 && gidz <= nz - npad - 3 - npml && gidx >= npml + 2 &&
      gidx <= nx - 3 - npml) {

    // update vx
    dsxx_dx = (c1 * (d_szz(gidz, gidx + 1) - d_szz(gidz, gidx)) -
               c2 * (d_szz(gidz, gidx + 2) - d_szz(gidz, gidx - 1))) / dx;
    d_vx(gidz, gidx) -= dsxx_dx * d_ave_Byc_b(gidz, gidx) * dt;

    // update vz
    dszz_dz = (c1 * (d_szz(gidz, gidx) - d_szz(gidz - 1, gidx)) -
               c2 * (d_szz(gidz + 1, gidx) - d_szz(gidz - 2, gidx))) / dz;
    d_vz(gidz, gidx) -= dszz_dz * d_ave_Byc_a(gidz, gidx) * dt;

  } else {
    return;
  }
}

__global__ void acoustic_backward_pressure(float *d_vz, float *d_vx,
                                           float *d_szz, float *d_Lambda,
                                           int nz, int nx, float dt, float dz,
                                           float dx, int npml, int npad) {

  int gidz = blockIdx.x * blockDim.x + threadIdx.x;
  int gidx = blockIdx.y * blockDim.y + threadIdx.y;

  float dvz_dz = 0.0;
  float dvx_dx = 0.0;

  float c1 = 9.0 / 8.0;
  float c2 = 1.0 / 24.0;

  // extension for derivative at the boundaries
  if (gidz >= npml + 2 && gidz <= nz - npad - 3 - npml && gidx >= npml + 2 &&
      gidx <= nx - 3 - npml) {
    // if (gidz>=npml-2 && gidz<=nz-npad+1-npml && gidx>=npml-2 &&
    // gidx<=nx+1-npml) {
    dvz_dz = (c1 * (d_vz(gidz + 1, gidx) - d_vz(gidz, gidx)) -
              c2 * (d_vz(gidz + 2, gidx) - d_vz(gidz - 1, gidx))) / dz;
    dvx_dx = (c1 * (d_vx(gidz, gidx) - d_vx(gidz, gidx - 1)) -
              c2 * (d_vx(gidz, gidx + 1) - d_vx(gidz, gidx - 2))) / dx;
    // d_mat_dvz_dz(gidz, gidx) = dvz_dz;
    // d_mat_dvx_dx(gidz, gidx) = dvx_dx;

    d_szz(gidz, gidx) -= d_Lambda(gidz, gidx) * (dvz_dz + dvx_dx) * dt;
  } else {
    return;
  }
}

__global__ void acoustic_adjoint_velocity(
    float *d_vz, float *d_vx, float *d_szz, float *d_mem_dvz_dz,
    float *d_mem_dvx_dx, float *d_mem_dszz_dz, float *d_mem_dsxx_dx,
    float *d_Lambda, float *d_ave_Byc_a, float *d_ave_Byc_b, float *d_K_z_half,
    float *d_a_z_half, float *d_b_z_half, float *d_K_x_half, float *d_a_x_half,
    float *d_b_x_half, float *d_K_z, float *d_a_z, float *d_b_z, float *d_K_x,
    float *d_a_x, float *d_b_x, int nz, int nx, float dt, float dz, float dx,
    int npml, int npad) {

  int gidz = blockIdx.x * blockDim.x + threadIdx.x;
  int gidx = blockIdx.y * blockDim.y + threadIdx.y;

  float dszz_dz = 0.0;
  float dsxx_dx = 0.0;
  float dpsiz_dz = 0.0;
  float dpsix_dx = 0.0;

  float c1 = 9.0 / 8.0;
  float c2 = 1.0 / 24.0;

  if (gidz >= 2 && gidz <= nz - npad - 3 && gidx >= 2 && gidx <= nx - 3) {

    dsxx_dx = (-c1 * (d_szz(gidz, gidx + 1) - d_szz(gidz, gidx)) +
               c2 * (d_szz(gidz, gidx + 2) - d_szz(gidz, gidx - 1))) / dx;
    dpsix_dx =
        (-c1 * (d_mem_dvx_dx(gidz, gidx + 1) - d_mem_dvx_dx(gidz, gidx)) +
         c2 * (d_mem_dvx_dx(gidz, gidx + 2) - d_mem_dvx_dx(gidz, gidx - 1))) /
        dx;
    
    d_vx(gidz, gidx) += (d_a_x[gidx] * dpsix_dx +
                         d_Lambda(gidz, gidx) * dsxx_dx / d_K_x[gidx] * dt);

    dszz_dz = (-c1 * (d_szz(gidz, gidx) - d_szz(gidz - 1, gidx)) +
               c2 * (d_szz(gidz + 1, gidx) - d_szz(gidz - 2, gidx))) /
              dz;
    dpsiz_dz =
        (-c1 * (d_mem_dvz_dz(gidz, gidx) - d_mem_dvz_dz(gidz - 1, gidx)) +
         c2 * (d_mem_dvz_dz(gidz + 1, gidx) - d_mem_dvz_dz(gidz - 2, gidx))) /
        dz;
    d_vz(gidz, gidx) +=
        (d_a_z_half[gidz] * dpsiz_dz +
         d_Lambda(gidz, gidx) * dszz_dz / d_K_z_half[gidz] * dt);

    // update psiz and psix
    if (gidx <= npml || gidx >= nx - npml - 1) {
      d_mem_dsxx_dx(gidz, gidx) =
          d_b_x_half[gidx] * d_mem_dsxx_dx(gidz, gidx) +
          d_ave_Byc_b(gidz, gidx) * d_vx(gidz, gidx) * dt;
    }
    if (gidz <= npml || (gidz >= nz - npml - npad - 1)) {
      d_mem_dszz_dz(gidz, gidx) =
          d_b_z[gidz] * d_mem_dszz_dz(gidz, gidx) +
          d_ave_Byc_a(gidz, gidx) * d_vz(gidz, gidx) * dt;
    }

  } else {
    return;
  }
}

__global__ void acoustic_adjoint_pressure(
    float *d_vz, float *d_vx, float *d_szz, float *d_mem_dvz_dz,
    float *d_mem_dvx_dx, float *d_mem_dszz_dz, float *d_mem_dsxx_dx,
    float *d_Lambda, float *d_ave_Byc_a, float *d_ave_Byc_b, float *d_K_z_half,
    float *d_a_z_half, float *d_b_z_half, float *d_K_x_half, float *d_a_x_half,
    float *d_b_x_half, float *d_K_z, float *d_a_z, float *d_b_z, float *d_K_x,
    float *d_a_x, float *d_b_x, int nz, int nx, float dt, float dz, float dx,
    int npml, int npad) {

  int gidz = blockIdx.x * blockDim.x + threadIdx.x;
  int gidx = blockIdx.y * blockDim.y + threadIdx.y;

  float dvz_dz = 0.0;
  float dvx_dx = 0.0;
  float dphiz_dz = 0.0;
  float dphix_dx = 0.0;

  float c1 = 9.0 / 8.0;
  float c2 = 1.0 / 24.0;

  if (gidz >= 2 && gidz <= nz - npad - 3 && gidx >= 2 && gidx <= nx - 3) {

    dvz_dz = (-c1 * (d_vz(gidz + 1, gidx) - d_vz(gidz, gidx)) +
              c2 * (d_vz(gidz + 2, gidx) - d_vz(gidz - 1, gidx))) /
             dz;
    dphiz_dz =
        (-c1 * (d_mem_dszz_dz(gidz + 1, gidx) - d_mem_dszz_dz(gidz, gidx)) +
         c2 * (d_mem_dszz_dz(gidz + 2, gidx) - d_mem_dszz_dz(gidz - 1, gidx))) /
        dz;

    // backward difference

    dvx_dx = (-c1 * (d_vx(gidz, gidx) - d_vx(gidz, gidx - 1)) +
              c2 * (d_vx(gidz, gidx + 1) - d_vx(gidz, gidx - 2))) /
             dx;
    dphix_dx =
        (-c1 * (d_mem_dsxx_dx(gidz, gidx) - d_mem_dsxx_dx(gidz, gidx - 1)) +
         c2 * (d_mem_dsxx_dx(gidz, gidx + 1) - d_mem_dsxx_dx(gidz, gidx - 2))) /
        dx;

    // update stress
    d_szz(gidz, gidx) +=
        d_a_x_half[gidx] * dphix_dx + d_a_z[gidz] * dphiz_dz +
        d_ave_Byc_b(gidz, gidx) * dvx_dx / d_K_x_half[gidx] * dt +
        d_ave_Byc_a(gidz, gidx) * dvz_dz / d_K_z[gidz] * dt;

    if (gidx <= npml || gidx >= nx - npml - 1) {
      d_mem_dvx_dx(gidz, gidx) = d_b_x[gidx] * d_mem_dvx_dx(gidz, gidx) +
                                 d_Lambda(gidz, gidx) * d_szz(gidz, gidx) * dt;
    }
    if (gidz <= npml || (gidz >= nz - npml - npad - 1)) {
      d_mem_dvz_dz(gidz, gidx) = d_b_z_half[gidz] * d_mem_dvz_dz(gidz, gidx) +
                                 d_Lambda(gidz, gidx) * d_szz(gidz, gidx) * dt;
    }

  } else {
    return;
  }
}

__global__ void image_vel_time(float *d_szz, float *d_szz_plusone,
                               float *d_szz_adj, int nz, int nx, float dt,
                               float dz, float dx, int npml, int npad,
                               float *d_Vp, float *d_Lambda, float *d_VpGrad) {

  int gidz = blockIdx.x * blockDim.x + threadIdx.x;
  int gidx = blockIdx.y * blockDim.y + threadIdx.y;

  // if (gidz>=2 && gidz<=nz-npad-3 && gidx>=2 && gidx<=nx-3) {
  if (gidz >= npml && gidz <= nz - npml - npad - 1 && gidx >= npml &&
      gidx <= nx - npml - 1) {

    d_szz_plusone(gidz, gidx) = (d_szz_plusone(gidz, gidx) - d_szz(gidz, gidx));
    d_VpGrad(gidz, gidx) += -2.0 / d_Vp(gidz, gidx) *
                            d_szz_plusone(gidz, gidx) * d_szz_adj(gidz, gidx);
  } else {
    return;
  }
}