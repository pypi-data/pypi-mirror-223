#include "hip/hip_runtime.h"
#define d_vx(z, x) d_vx[(x) * (nz) + (z)]
#define d_vz(z, x) d_vz[(x) * (nz) + (z)]
#define d_sxx(z, x) d_sxx[(x) * (nz) + (z)]
#define d_szz(z, x) d_szz[(x) * (nz) + (z)]
#define d_sxz(z, x) d_sxz[(x) * (nz) + (z)]
#define d_vx_adj(z, x) d_vx_adj[(x) * (nz) + (z)]
#define d_vz_adj(z, x) d_vz_adj[(x) * (nz) + (z)]
#define d_sxx_adj(z, x) d_sxx_adj[(x) * (nz) + (z)]
#define d_szz_adj(z, x) d_szz_adj[(x) * (nz) + (z)]
#define d_sxz_adj(z, x) d_sxz_adj[(x) * (nz) + (z)]
#define d_mem_dszz_dz(z, x) d_mem_dszz_dz[(x) * (nz) + (z)]
#define d_mem_dsxz_dx(z, x) d_mem_dsxz_dx[(x) * (nz) + (z)]
#define d_mem_dsxz_dz(z, x) d_mem_dsxz_dz[(x) * (nz) + (z)]
#define d_mem_dsxx_dx(z, x) d_mem_dsxx_dx[(x) * (nz) + (z)]
#define d_mem_dvz_dz(z, x) d_mem_dvz_dz[(x) * (nz) + (z)]
#define d_mem_dvz_dx(z, x) d_mem_dvz_dx[(x) * (nz) + (z)]
#define d_mem_dvx_dz(z, x) d_mem_dvx_dz[(x) * (nz) + (z)]
#define d_mem_dvx_dx(z, x) d_mem_dvx_dx[(x) * (nz) + (z)]
#define d_Lambda(z, x) d_Lambda[(x) * (nz) + (z)]
#define d_Mu(z, x) d_Mu[(x) * (nz) + (z)]
#define d_Den(z, x) d_Den[(x) * (nz) + (z)]
#define d_ave_Mu(z, x) d_ave_Mu[(x) * (nz) + (z)]
#define d_ave_Byc_a(z, x) d_ave_Byc_a[(x) * (nz) + (z)]
#define d_ave_Byc_b(z, x) d_ave_Byc_b[(x) * (nz) + (z)]
#define d_LambdaGrad(z, x) d_LambdaGrad[(x) * (nz) + (z)]
#define d_MuGrad(z, x) d_MuGrad[(x) * (nz) + (z)]
#define d_DenGrad(z, x) d_DenGrad[(x) * (nz) + (z)]
#include "utils.h"
#include <stdio.h>

// elastic_forward_velocity
// elastic_forward_stress
// elastic_backward_stress
// elastic_backward_velocity
// elastic_forward_stress
// elastic_forward_velocity

// elastic forward modeling: update velocity
__global__ void elastic_forward_velocity(
    float *d_vz, float *d_vx, float *d_szz, float *d_sxx, float *d_sxz,
    float *d_mem_dszz_dz, float *d_mem_dsxz_dx, float *d_mem_dsxz_dz,
    float *d_mem_dsxx_dx, float *d_ave_Byc_a, float *d_ave_Byc_b, float *d_K_z,
    float *d_a_z, float *d_b_z, float *d_K_z_half, float *d_a_z_half,
    float *d_b_z_half, float *d_K_x, float *d_a_x, float *d_b_x,
    float *d_K_x_half, float *d_a_x_half, float *d_b_x_half, int nz, int nx,
    float dt, float dz, float dx, int npml, int npad) {

  // get the global grid index
  int gidz = blockIdx.x * blockDim.x + threadIdx.x;
  int gidx = blockIdx.y * blockDim.y + threadIdx.y;

  // initialize the memory
  float dszz_dz = 0.0;
  float dsxz_dx = 0.0;
  float dsxz_dz = 0.0;
  float dsxx_dx = 0.0;

  // 4th-order FD coefficients
  float c1 = 9.0 / 8.0;
  float c2 = 1.0 / 24.0;

  // update velocity
  if (gidz >= 2 && gidz <= nz - npad - 3 && gidx >= 2 && gidx <= nx - 3) {
    // derivative of stress
    dszz_dz = (c1 * (d_szz(gidz + 1, gidx) - d_szz(gidz, gidx)) -
               c2 * (d_szz(gidz + 2, gidx) - d_szz(gidz - 1, gidx))) / dz;
    dsxz_dx = (c1 * (d_sxz(gidz, gidx) - d_sxz(gidz, gidx - 1)) -
               c2 * (d_sxz(gidz, gidx + 1) - d_sxz(gidz, gidx - 2))) / dx;

    // pml boundary
    if (gidz < npml || (gidz > nz - npml - npad - 1)) {
      d_mem_dszz_dz(gidz, gidx) = d_b_z_half[gidz] * d_mem_dszz_dz(gidz, gidx) +
                                  d_a_z_half[gidz] * dszz_dz;
      dszz_dz = dszz_dz / d_K_z_half[gidz] + d_mem_dszz_dz(gidz, gidx);
    }
    // pml boundary
    if (gidx < npml || gidx > nx - npml) {
      d_mem_dsxz_dx(gidz, gidx) =
          d_b_x[gidx] * d_mem_dsxz_dx(gidz, gidx) + d_a_x[gidx] * dsxz_dx;
      dsxz_dx = dsxz_dx / d_K_x[gidx] + d_mem_dsxz_dx(gidz, gidx);
    }
    // update vz
    d_vz(gidz, gidx) += (dszz_dz + dsxz_dx) * d_ave_Byc_a(gidz, gidx) * dt;

    // derivative of stress
    dsxz_dz = (c1 * (d_sxz(gidz, gidx) - d_sxz(gidz - 1, gidx)) -
               c2 * (d_sxz(gidz + 1, gidx) - d_sxz(gidz - 2, gidx))) / dz;
    dsxx_dx = (c1 * (d_sxx(gidz, gidx + 1) - d_sxx(gidz, gidx)) -
               c2 * (d_sxx(gidz, gidx + 2) - d_sxx(gidz, gidx - 1))) / dx;
    // pml boundary
    if (gidz < npml || (gidz > nz - npml - npad - 1)) {
      d_mem_dsxz_dz(gidz, gidx) = d_b_z[gidz] * d_mem_dsxz_dz(gidz, gidx) + 
                                  d_a_z[gidz] * dsxz_dz;
      dsxz_dz = dsxz_dz / d_K_z[gidz] + d_mem_dsxz_dz(gidz, gidx);
    }
    // pml boundary
    if (gidx < npml || gidx > nx - npml) {
      d_mem_dsxx_dx(gidz, gidx) = d_b_x_half[gidx] * d_mem_dsxx_dx(gidz, gidx) +
                                  d_a_x_half[gidx] * dsxx_dx;
      dsxx_dx = dsxx_dx / d_K_x_half[gidx] + d_mem_dsxx_dx(gidz, gidx);
    }
    // update vx
    d_vx(gidz, gidx) += (dsxz_dz + dsxx_dx) * d_ave_Byc_b(gidz, gidx) * dt;

  } else {
    return;
  }
}

// elastic forward modeling: update stress
__global__ void elastic_forward_stress(
    float *d_vz, float *d_vx, float *d_szz, float *d_sxx, float *d_sxz,
    float *d_mem_dvz_dz, float *d_mem_dvz_dx, float *d_mem_dvx_dz,
    float *d_mem_dvx_dx, float *d_Lambda, float *d_Mu, float *d_ave_Mu,
    float *d_K_z, float *d_a_z, float *d_b_z, float *d_K_z_half,
    float *d_a_z_half, float *d_b_z_half, float *d_K_x, float *d_a_x,
    float *d_b_x, float *d_K_x_half, float *d_a_x_half, float *d_b_x_half,
    int nz, int nx, float dt, float dz, float dx, int npml, int npad) {

  // calculate the global index
  int gidz = blockIdx.x * blockDim.x + threadIdx.x;
  int gidx = blockIdx.y * blockDim.y + threadIdx.y;

  // initialize the derivatives
  float dvz_dz = 0.0;
  float dvx_dx = 0.0;
  float dvx_dz = 0.0;
  float dvz_dx = 0.0;

  // 4th-order FD coefficients
  float c1 = 9.0 / 8.0;
  float c2 = 1.0 / 24.0;

  if (gidz >= 2 && gidz <= nz - npad - 3 && gidx >= 2 && gidx <= nx - 3) {

    // update sxx and szz
    dvz_dz = (c1 * (d_vz(gidz, gidx) - d_vz(gidz - 1, gidx)) -
              c2 * (d_vz(gidz + 1, gidx) - d_vz(gidz - 2, gidx))) / dz;
    dvx_dx = (c1 * (d_vx(gidz, gidx) - d_vx(gidz, gidx - 1)) -
              c2 * (d_vx(gidz, gidx + 1) - d_vx(gidz, gidx - 2))) / dx;

    if (gidz < npml || (gidz > nz - npml - npad - 1)) {
      d_mem_dvz_dz(gidz, gidx) = d_b_z[gidz] * d_mem_dvz_dz(gidz, gidx) + 
                                 d_a_z[gidz] * dvz_dz;
      dvz_dz = dvz_dz / d_K_z[gidz] + d_mem_dvz_dz(gidz, gidx);
    }
    if (gidx < npml || gidx > nx - npml - 1) {
      d_mem_dvx_dx(gidz, gidx) = d_b_x[gidx] * d_mem_dvx_dx(gidz, gidx) + 
                                 d_a_x[gidx] * dvx_dx;
      dvx_dx = dvx_dx / d_K_x[gidx] + d_mem_dvx_dx(gidz, gidx);
    }

    d_szz(gidz, gidx) +=
        ((d_Lambda(gidz, gidx) + 2.0 * d_Mu(gidz, gidx)) * dvz_dz +
          d_Lambda(gidz, gidx) * dvx_dx) * dt;
    d_sxx(gidz, gidx) +=
        (d_Lambda(gidz, gidx) * dvz_dz +
        (d_Lambda(gidz, gidx) + 2.0 * d_Mu(gidz, gidx)) * dvx_dx) * dt;

    // update sxz
    dvx_dz = (c1 * (d_vx(gidz + 1, gidx) - d_vx(gidz, gidx)) -
              c2 * (d_vx(gidz + 2, gidx) - d_vx(gidz - 1, gidx))) / dz;
    dvz_dx = (c1 * (d_vz(gidz, gidx + 1) - d_vz(gidz, gidx)) -
              c2 * (d_vz(gidz, gidx + 2) - d_vz(gidz, gidx - 1))) / dx;

    if (gidz < npml || (gidz > nz - npml - npad - 1)) {
      d_mem_dvx_dz(gidz, gidx) = d_b_z_half[gidz] * d_mem_dvx_dz(gidz, gidx) +
                                 d_a_z_half[gidz] * dvx_dz;
      dvx_dz = dvx_dz / d_K_z_half[gidz] + d_mem_dvx_dz(gidz, gidx);
    }
    if (gidx < npml || gidx > nx - npml - 1) {
      d_mem_dvz_dx(gidz, gidx) = d_b_x_half[gidx] * d_mem_dvz_dx(gidz, gidx) +
                                 d_a_x_half[gidx] * dvz_dx;
      dvz_dx = dvz_dx / d_K_x_half[gidx] + d_mem_dvz_dx(gidz, gidx);
    }

    d_sxz(gidz, gidx) += d_ave_Mu(gidz, gidx) * (dvx_dz + dvz_dx) * dt;

  } else {
    return;
  }
}

// elastic backward modeling: update velocity
__global__ void elastic_backward_velocity(float *d_vz, float *d_vx,
                                          float *d_szz, float *d_sxx,
                                          float *d_sxz, float *d_ave_Byc_a,
                                          float *d_ave_Byc_b, int nz, int nx,
                                          float dt, float dz, float dx,
                                          int npml, int npad, float *d_vz_adj,
                                          float *d_vx_adj, float *d_DenGrad) {

  int gidz = blockIdx.x * blockDim.x + threadIdx.x;
  int gidx = blockIdx.y * blockDim.y + threadIdx.y;

  float dszz_dz = 0.0;
  float dsxz_dx = 0.0;
  float dsxz_dz = 0.0;
  float dsxx_dx = 0.0;

  float c1 = 9.0 / 8.0;
  float c2 = 1.0 / 24.0;

  if (gidz >= npml && gidz <= nz - npad - 1 - npml && gidx >= npml &&
      gidx <= nx - 1 - npml) {
    // update vz
    dszz_dz = (c1 * (d_szz(gidz + 1, gidx) - d_szz(gidz, gidx)) -
               c2 * (d_szz(gidz + 2, gidx) - d_szz(gidz - 1, gidx))) / dz;
    dsxz_dx = (c1 * (d_sxz(gidz, gidx) - d_sxz(gidz, gidx - 1)) -
               c2 * (d_sxz(gidz, gidx + 1) - d_sxz(gidz, gidx - 2))) / dx;

    d_vz(gidz, gidx) -= (dszz_dz + dsxz_dx) * d_ave_Byc_a(gidz, gidx) * dt;

    // update vx
    dsxz_dz = (c1 * (d_sxz(gidz, gidx) - d_sxz(gidz - 1, gidx)) -
               c2 * (d_sxz(gidz + 1, gidx) - d_sxz(gidz - 2, gidx))) / dz;
    dsxx_dx = (c1 * (d_sxx(gidz, gidx + 1) - d_sxx(gidz, gidx)) -
               c2 * (d_sxx(gidz, gidx + 2) - d_sxx(gidz, gidx - 1))) / dx;

    d_vx(gidz, gidx) -= (dsxz_dz + dsxx_dx) * d_ave_Byc_b(gidz, gidx) * dt;

    // cross-correlation image condition: density kernel (spray)
    float grad_ave_Byc_a = -d_vz_adj(gidz, gidx) * (dszz_dz + dsxz_dx) * dt *
                           (-pow(d_ave_Byc_a(gidz, gidx), 2) / 2.0);
    float grad_ave_Byc_b = -d_vx_adj(gidz, gidx) * (dsxz_dz + dsxx_dx) * dt *
                           (-pow(d_ave_Byc_b(gidz, gidx), 2) / 2.0);
    atomicAdd(&d_DenGrad[gidz + nz * gidx], grad_ave_Byc_a);
    atomicAdd(&d_DenGrad[gidz + nz * gidx], grad_ave_Byc_b);
    if (gidz + 1 <= nz - npad - 1 - npml) {
      atomicAdd(&d_DenGrad[gidz + 1 + nz * gidx], grad_ave_Byc_a);
    }
    if (gidx + 1 <= gidx <= nx - 1 - npml) {
      atomicAdd(&d_DenGrad[gidz + nz * (gidx + 1)], grad_ave_Byc_b);
    }
  } else {
    return;
  }
}

// elastic backward modeling: update stress
__global__ void elastic_backward_stress(
    float *d_vz, float *d_vx, float *d_szz, float *d_sxx, float *d_sxz,
    float *d_Lambda, float *d_Mu, float *d_ave_Mu, int nz, int nx, float dt,
    float dz, float dx, int npml, int npad, float *d_szz_adj, float *d_sxx_adj,
    float *d_sxz_adj, float *d_LambdaGrad, float *d_MuGrad) {

  int gidz = blockIdx.x * blockDim.x + threadIdx.x;
  int gidx = blockIdx.y * blockDim.y + threadIdx.y;

  float dvz_dz = 0.0;
  float dvx_dx = 0.0;
  float dvx_dz = 0.0;
  float dvz_dx = 0.0;

  float c1 = 9.0 / 8.0;
  float c2 = 1.0 / 24.0;

  if (gidz >= npml && gidz <= nz - npad - 1 - npml && gidx >= npml &&
      gidx <= nx - 1 - npml) {

    dvz_dz = (c1 * (d_vz(gidz, gidx) - d_vz(gidz - 1, gidx)) -
              c2 * (d_vz(gidz + 1, gidx) - d_vz(gidz - 2, gidx))) / dz;
    dvx_dx = (c1 * (d_vx(gidz, gidx) - d_vx(gidz, gidx - 1)) -
              c2 * (d_vx(gidz, gidx + 1) - d_vx(gidz, gidx - 2))) / dx;

    d_szz(gidz, gidx) -=
        ((d_Lambda(gidz, gidx) + 2.0 * d_Mu(gidz, gidx)) * dvz_dz +
         d_Lambda(gidz, gidx) * dvx_dx) * dt;
    d_sxx(gidz, gidx) -=
        (d_Lambda(gidz, gidx) * dvz_dz +
         (d_Lambda(gidz, gidx) + 2.0 * d_Mu(gidz, gidx)) * dvx_dx) * dt;

    dvx_dz = (c1 * (d_vx(gidz + 1, gidx) - d_vx(gidz, gidx)) -
              c2 * (d_vx(gidz + 2, gidx) - d_vx(gidz - 1, gidx))) / dz;
    dvz_dx = (c1 * (d_vz(gidz, gidx + 1) - d_vz(gidz, gidx)) -
              c2 * (d_vz(gidz, gidx + 2) - d_vz(gidz, gidx - 1))) / dx;

    d_sxz(gidz, gidx) -= d_ave_Mu(gidz, gidx) * (dvx_dz + dvz_dx) * dt;

    // cross-correlation image condition: lambda and mu kernels (spray)
    d_LambdaGrad(gidz, gidx) +=
        -(d_szz_adj(gidz, gidx) + d_sxx_adj(gidz, gidx)) * (dvz_dz + dvx_dx) *
        dt * MEGA;
    d_MuGrad(gidz, gidx) += (-2.0 * d_szz_adj(gidz, gidx) * dvz_dz * dt -
                             2.0 * d_sxx_adj(gidz, gidx) * dvx_dx * dt) * MEGA;

    // spray (atomic add)
    if (d_ave_Mu(gidz, gidx) != 0.0) {
      float scale =
          -d_sxz_adj(gidz, gidx) * (dvx_dz + dvz_dx) * dt *
          d_ave_Mu(gidz, gidx) /
          (1.0 / d_Mu(gidz, gidx) + 1.0 / d_Mu(gidz + 1, gidx) +
           1.0 / d_Mu(gidz, gidx + 1) + 1.0 / d_Mu(gidz + 1, gidx + 1)) * MEGA;
      atomicAdd(&d_MuGrad[gidz + nz * gidx],
                1.0 / pow(d_Mu(gidz, gidx), 2) * scale);
      if (gidz + 1 <= nz - npad - 1 - npml) {
        atomicAdd(&d_MuGrad[gidz + 1 + nz * gidx],
                  1.0 / pow(d_Mu(gidz + 1, gidx), 2) * scale);
      }
      if (gidx + 1 <= gidx <= nx - 1 - npml) {
        atomicAdd(&d_MuGrad[gidz + nz * (gidx + 1)],
                  1.0 / pow(d_Mu(gidz, gidx + 1), 2) * scale);
      }
      if (gidz + 1 <= nz - npad - 1 - npml && gidx + 1 <= nx - 1 - npml) {
        atomicAdd(&d_MuGrad[gidz + 1 + nz * (gidx + 1)],
                  1.0 / pow(d_Mu(gidz + 1, gidx + 1), 2) * scale);
      }
    }
  } else {
    return;
  }
}


// elastic adjoint modeling: update velocity
__global__ void elastic_adjoint_velocity(
  float *d_vz, float *d_vx, float *d_szz, float *d_sxx, float *d_sxz,
  float *d_mem_dszz_dz, float *d_mem_dsxz_dx, float *d_mem_dsxz_dz,
  float *d_mem_dsxx_dx, float *d_mem_dvz_dz, float *d_mem_dvz_dx,
  float *d_mem_dvx_dz, float *d_mem_dvx_dx, float *d_Lambda, float *d_Mu,
  float *d_ave_Mu, float *d_Den, float *d_ave_Byc_a, float *d_ave_Byc_b,
  float *d_K_z_half, float *d_a_z_half, float *d_b_z_half, float *d_K_x_half,
  float *d_a_x_half, float *d_b_x_half, float *d_K_z, float *d_a_z,
  float *d_b_z, float *d_K_x, float *d_a_x, float *d_b_x, int nz, int nx,
  float dt, float dz, float dx, int npml, int npad) {

  int gidz = blockIdx.x * blockDim.x + threadIdx.x;
  int gidx = blockIdx.y * blockDim.y + threadIdx.y;

  float dpsixx_dx = 0.0;
  float dszz_dx = 0.0;
  float dsxx_dx = 0.0;
  float dpsixz_dz = 0.0;
  float dsxz_dz = 0.0;
  float dpsizz_dz = 0.0;
  float dszz_dz = 0.0;
  float dsxx_dz = 0.0;
  float dpsizx_dx = 0.0;
  float dsxz_dx = 0.0;

  float c1 = 9.0 / 8.0;
  float c2 = 1.0 / 24.0;

  // float lambda = d_Lambda(gidz, gidx);
  // float mu = d_Mu(gidz, gidx);

  if (gidz >= 2 && gidz <= nz - npad - 3 && gidx >= 2 && gidx <= nx - 3) {

    // update vx
    dpsixx_dx =
        (-c1 * (d_mem_dvx_dx(gidz, gidx + 1) - d_mem_dvx_dx(gidz, gidx)) +
        c2 * (d_mem_dvx_dx(gidz, gidx + 2) - d_mem_dvx_dx(gidz, gidx - 1))) / dx;

    dszz_dx = (-c1 * (d_szz(gidz, gidx + 1) * d_Lambda(gidz, gidx + 1) - 
                      d_szz(gidz, gidx    ) * d_Lambda(gidz, gidx   )) +
                c2 * (d_szz(gidz, gidx + 2) * d_Lambda(gidz, gidx + 2) - 
                      d_szz(gidz, gidx - 1) * d_Lambda(gidz, gidx - 1))) / dx;

    dsxx_dx = (-c1 * (d_sxx(gidz, gidx + 1) * (d_Lambda(gidz, gidx + 1) + 2.0 * d_Mu(gidz, gidx + 1)) -
                      d_sxx(gidz, gidx    ) * (d_Lambda(gidz, gidx    ) + 2.0 * d_Mu(gidz, gidx    ))) +
                c2 * (d_sxx(gidz, gidx + 2) * (d_Lambda(gidz, gidx + 2) + 2.0 * d_Mu(gidz, gidx + 2)) - 
                      d_sxx(gidz, gidx - 1) * (d_Lambda(gidz, gidx - 1) + 2.0 * d_Mu(gidz, gidx - 1)))) / dx;

    dpsixz_dz =
        (-c1 * (d_mem_dvx_dz(gidz, gidx) - d_mem_dvx_dz(gidz - 1, gidx)) +
        c2 * (d_mem_dvx_dz(gidz + 1, gidx) - d_mem_dvx_dz(gidz - 2, gidx))) / dz;

    dsxz_dz = (-c1 * (d_sxz(gidz    , gidx) * d_ave_Mu(gidz    , gidx) - 
                      d_sxz(gidz - 1, gidx) * d_ave_Mu(gidz - 1, gidx)) +
                c2 * (d_sxz(gidz + 1, gidx) * d_ave_Mu(gidz + 1, gidx) - 
                      d_sxz(gidz - 2, gidx) * d_ave_Mu(gidz - 2, gidx))) / dz;

    d_vx(gidz, gidx) +=
        (d_a_x[gidx] * dpsixx_dx + 
        d_a_z_half[gidz] * dpsixz_dz + 
        dszz_dx / d_K_x[gidx] * dt + 
        dsxx_dx / d_K_x[gidx] * dt +
        dsxz_dz / d_K_z_half[gidz] * dt);


    // update phi_xx_x and phi_xz_z
    if (gidx < npml || gidx > nx - npml - 1) {
      d_mem_dsxx_dx(gidz, gidx) =
          d_b_x_half[gidx] * d_mem_dsxx_dx(gidz, gidx) +
          d_ave_Byc_b(gidz, gidx) * d_vx(gidz, gidx) * dt;
    }
    if (gidz < npml || (gidz > nz - npml - npad - 1)) {
      d_mem_dsxz_dz(gidz, gidx) =
          d_b_z[gidz] * d_mem_dsxz_dz(gidz, gidx) +
          d_ave_Byc_b(gidz, gidx) * d_vx(gidz, gidx) * dt;
    }

    // update vz
    dpsizz_dz =
        (-c1 * (d_mem_dvz_dz(gidz + 1, gidx) - d_mem_dvz_dz(gidz, gidx)) +
        c2 * (d_mem_dvz_dz(gidz + 2, gidx) - d_mem_dvz_dz(gidz - 1, gidx))) / dz;

    dszz_dz = (-c1 * (d_szz(gidz + 1, gidx) * (d_Lambda(gidz + 1, gidx) + 2.0 * d_Mu(gidz + 1, gidx)) - 
                      d_szz(gidz    , gidx) * (d_Lambda(gidz    , gidx) + 2.0 * d_Mu(gidz    , gidx))) +
                c2 * (d_szz(gidz + 2, gidx) * (d_Lambda(gidz + 2, gidx) + 2.0 * d_Mu(gidz + 2, gidx)) - 
                      d_szz(gidz - 1, gidx) * (d_Lambda(gidz - 1, gidx) + 2.0 * d_Mu(gidz - 1, gidx)))) / dz;

    dsxx_dz = (-c1 * (d_sxx(gidz + 1, gidx) * d_Lambda(gidz + 1, gidx) - 
                      d_sxx(gidz    , gidx) * d_Lambda(gidz    , gidx)) +
                c2 * (d_sxx(gidz + 2, gidx) * d_Lambda(gidz + 2, gidx) - 
                      d_sxx(gidz - 1, gidx) * d_Lambda(gidz - 1, gidx))) / dz;

    dpsizx_dx =
        (-c1 * (d_mem_dvz_dx(gidz, gidx) - d_mem_dvz_dx(gidz, gidx - 1)) +
        c2 * (d_mem_dvz_dx(gidz, gidx + 1) - d_mem_dvz_dx(gidz, gidx - 2))) / dx;

    dsxz_dx = (-c1 * (d_sxz(gidz, gidx    ) * d_ave_Mu(gidz, gidx    ) - 
                      d_sxz(gidz, gidx - 1) * d_ave_Mu(gidz, gidx - 1)) +
                c2 * (d_sxz(gidz, gidx + 1) * d_ave_Mu(gidz, gidx + 1) - 
                      d_sxz(gidz, gidx - 2) * d_ave_Mu(gidz, gidx - 2))) / dx;

    d_vz(gidz, gidx) +=
        (d_a_z[gidz] * dpsizz_dz + 
        dszz_dz / d_K_z[gidz] * dt +
        dsxx_dz / d_K_z[gidz] * dt + 
        d_a_x_half[gidx] * dpsizx_dx +
        dsxz_dx / d_K_x_half[gidx] * dt);

    // update phi_xz_x and phi_zz_z
    if (gidx < npml || gidx > nx - npml - 1) {
      d_mem_dsxz_dx(gidz, gidx) =
          d_b_x[gidx] * d_mem_dsxz_dx(gidz, gidx) +
          d_ave_Byc_a(gidz, gidx) * d_vz(gidz, gidx) * dt;
    }
    if (gidz < npml || (gidz > nz - npml - npad - 1)) {
      d_mem_dszz_dz(gidz, gidx) =
          d_b_z_half[gidz] * d_mem_dszz_dz(gidz, gidx) +
          d_ave_Byc_a(gidz, gidx) * d_vz(gidz, gidx) * dt;
    }

  }

  else {
    return;
  }
}


// elastic adjoint modeling: update stress
__global__ void elastic_adjoint_stress(
  float *d_vz, float *d_vx, float *d_szz, float *d_sxx, float *d_sxz,
  float *d_mem_dszz_dz, float *d_mem_dsxz_dx, float *d_mem_dsxz_dz,
  float *d_mem_dsxx_dx, float *d_mem_dvz_dz, float *d_mem_dvz_dx,
  float *d_mem_dvx_dz, float *d_mem_dvx_dx, float *d_Lambda, float *d_Mu,
  float *d_ave_Mu, float *d_Den, float *d_ave_Byc_a, float *d_ave_Byc_b,
  float *d_K_z_half, float *d_a_z_half, float *d_b_z_half, float *d_K_x_half,
  float *d_a_x_half, float *d_b_x_half, float *d_K_z, float *d_a_z,
  float *d_b_z, float *d_K_x, float *d_a_x, float *d_b_x, int nz, int nx,
  float dt, float dz, float dx, int npml, int npad) {

  int gidz = blockIdx.x * blockDim.x + threadIdx.x;
  int gidx = blockIdx.y * blockDim.y + threadIdx.y;

  float dphi_xz_x_dx = 0.0;
  float dvz_dx = 0.0;
  float dphi_xz_z_dz = 0.0;
  float dvx_dz = 0.0;
  float dphi_xx_x_dx = 0.0;
  float dvx_dx = 0.0;
  float dphi_zz_z_dz = 0.0;
  float dvz_dz = 0.0;

  float c1 = 9.0 / 8.0;
  float c2 = 1.0 / 24.0;

  float lambda = d_Lambda(gidz, gidx);
  float mu = d_Mu(gidz, gidx);

  if (gidz >= 2 && gidz <= nz - npad - 3 && gidx >= 2 && gidx <= nx - 3) {

    dphi_xz_x_dx =
        (-c1 * (d_mem_dsxz_dx(gidz, gidx + 1) - d_mem_dsxz_dx(gidz, gidx)) +
          c2 * (d_mem_dsxz_dx(gidz, gidx + 2) - d_mem_dsxz_dx(gidz, gidx - 1))) / dx;

    dvz_dx = (-c1 * (d_vz(gidz, gidx + 1) * d_ave_Byc_a(gidz, gidx + 1) - 
                    d_vz(gidz, gidx    ) * d_ave_Byc_a(gidz, gidx    )) +
              c2 * (d_vz(gidz, gidx + 2) * d_ave_Byc_a(gidz, gidx + 2) - 
                    d_vz(gidz, gidx - 1) * d_ave_Byc_a(gidz, gidx - 1))) / dx;

    dphi_xz_z_dz =
        (-c1 * (d_mem_dsxz_dz(gidz + 1, gidx) - d_mem_dsxz_dz(gidz, gidx)) +
        c2 * (d_mem_dsxz_dz(gidz + 2, gidx) - d_mem_dsxz_dz(gidz - 1, gidx))) / dz;

    dvx_dz = (-c1 * (d_vx(gidz + 1, gidx) * d_ave_Byc_b(gidz + 1, gidx) - 
                    d_vx(gidz    , gidx) * d_ave_Byc_b(gidz    , gidx)) +
              c2 * (d_vx(gidz + 2, gidx) * d_ave_Byc_b(gidz + 2, gidx) - 
                    d_vx(gidz - 1, gidx) * d_ave_Byc_b(gidz - 1, gidx))) / dz;

    // update sxz
    d_sxz(gidz, gidx) += d_a_x[gidx] * dphi_xz_x_dx +
                        dvz_dx / d_K_x[gidx] * dt +
                        d_a_z[gidz] * dphi_xz_z_dz +
                        dvx_dz / d_K_z[gidz] * dt;

    // update psi_zx and psi_xz
    // if(gidx<npml || gidx>nx-npml-1){
    d_mem_dvz_dx(gidz, gidx) = d_b_x_half[gidx] * d_mem_dvz_dx(gidz, gidx) +
                              d_sxz(gidz, gidx) * d_ave_Mu(gidz, gidx) * dt;
    // }
    // if(gidz<npml || gidz>nz-npml-npad-1){
    d_mem_dvx_dz(gidz, gidx) = d_b_z_half[gidz] * d_mem_dvx_dz(gidz, gidx) +
                              d_sxz(gidz, gidx) * d_ave_Mu(gidz, gidx) * dt;
    // }

    dphi_xx_x_dx =
        (-c1 * (d_mem_dsxx_dx(gidz, gidx) - d_mem_dsxx_dx(gidz, gidx - 1)) +
        c2 * (d_mem_dsxx_dx(gidz, gidx + 1) - d_mem_dsxx_dx(gidz, gidx - 2))) / dx;

    dvx_dx = (-c1 * (d_vx(gidz, gidx    ) * d_ave_Byc_b(gidz, gidx    ) - 
                    d_vx(gidz, gidx - 1) * d_ave_Byc_b(gidz, gidx - 1)) +
              c2 * (d_vx(gidz, gidx + 1) * d_ave_Byc_b(gidz, gidx + 1) - 
                    d_vx(gidz, gidx - 2) * d_ave_Byc_b(gidz, gidx - 2))) / dx;

    dphi_zz_z_dz =
        (-c1 * (d_mem_dszz_dz(gidz, gidx) - d_mem_dszz_dz(gidz - 1, gidx)) +
        c2 * (d_mem_dszz_dz(gidz + 1, gidx) - d_mem_dszz_dz(gidz - 2, gidx))) / dz;

    dvz_dz = (-c1 * (d_vz(gidz    , gidx) * d_ave_Byc_a(gidz    , gidx) - 
                    d_vz(gidz - 1, gidx) * d_ave_Byc_a(gidz - 1, gidx)) +
              c2 * (d_vz(gidz + 1, gidx) * d_ave_Byc_a(gidz + 1, gidx) - 
                    d_vz(gidz - 2, gidx) * d_ave_Byc_a(gidz - 2, gidx))) / dz;

    // update sxx and szz
    d_sxx(gidz, gidx) += d_a_x_half[gidx] * dphi_xx_x_dx + dvx_dx / d_K_x_half[gidx] * dt;

    d_szz(gidz, gidx) += d_a_z_half[gidz] * dphi_zz_z_dz + dvz_dz / d_K_z_half[gidz] * dt;

    // update psi_xx and psi_zz
    // if(gidx<npml || gidx>nx-npml-1){
    d_mem_dvx_dx(gidz, gidx) = d_b_x[gidx] * d_mem_dvx_dx(gidz, gidx) +
                              lambda * d_szz(gidz, gidx) * dt +
                              (lambda + 2.0 * mu) * d_sxx(gidz, gidx) * dt;
    // }
    // if(gidz<npml || (gidz>nz-npml-npad-1)){
    d_mem_dvz_dz(gidz, gidx) = d_b_z[gidz] * d_mem_dvz_dz(gidz, gidx) +
                              (lambda + 2.0 * mu) * d_szz(gidz, gidx) * dt +
                              lambda * d_sxx(gidz, gidx) * dt;
    // }

  } else {
    return;
  }
}



// // elastic adjoint modeling: update velocity
// __global__ void elastic_adjoint_velocity(
//     float *d_vz, float *d_vx, float *d_szz, float *d_sxx, float *d_sxz,
//     float *d_mem_dszz_dz, float *d_mem_dsxz_dx, float *d_mem_dsxz_dz,
//     float *d_mem_dsxx_dx, float *d_mem_dvz_dz, float *d_mem_dvz_dx,
//     float *d_mem_dvx_dz, float *d_mem_dvx_dx, float *d_Lambda, float *d_Mu,
//     float *d_ave_Mu, float *d_Den, float *d_ave_Byc_a, float *d_ave_Byc_b,
//     float *d_K_z_half, float *d_a_z_half, float *d_b_z_half, float *d_K_x_half,
//     float *d_a_x_half, float *d_b_x_half, float *d_K_z, float *d_a_z,
//     float *d_b_z, float *d_K_x, float *d_a_x, float *d_b_x, int nz, int nx,
//     float dt, float dz, float dx, int npml, int npad) {

//   int gidz = blockIdx.x * blockDim.x + threadIdx.x;
//   int gidx = blockIdx.y * blockDim.y + threadIdx.y;

//   float dpsixx_dx = 0.0;
//   float dszz_dx = 0.0;
//   float dsxx_dx = 0.0;
//   float dpsixz_dz = 0.0;
//   float dsxz_dz = 0.0;
//   float dpsizz_dz = 0.0;
//   float dszz_dz = 0.0;
//   float dsxx_dz = 0.0;
//   float dpsizx_dx = 0.0;
//   float dsxz_dx = 0.0;

//   float c1 = 9.0 / 8.0;
//   float c2 = 1.0 / 24.0;

//   float lambda = d_Lambda(gidz, gidx);
//   float mu = d_Mu(gidz, gidx);

//   if (gidz >= 2 && gidz <= nz - npad - 3 && gidx >= 2 && gidx <= nx - 3) {

// 		// update vx
// 		dpsixx_dx = (-c1*(d_mem_dvx_dx(gidz,gidx+1)-d_mem_dvx_dx(gidz,gidx)) \
// 			         	+ c2*(d_mem_dvx_dx(gidz,gidx+2)-d_mem_dvx_dx(gidz,gidx-1)))/dx;
// 		dszz_dx = (-c1*(d_szz(gidz,gidx+1)-d_szz(gidz,gidx)) + c2*(d_szz(gidz,gidx+2)-d_szz(gidz,gidx-1)))/dx;
// 		dsxx_dx = (-c1*(d_sxx(gidz,gidx+1)-d_sxx(gidz,gidx)) + c2*(d_sxx(gidz,gidx+2)-d_sxx(gidz,gidx-1)))/dx;
// 		dpsixz_dz = (-c1*(d_mem_dvx_dz(gidz,gidx)-d_mem_dvx_dz(gidz-1,gidx)) \
// 				        + c2*(d_mem_dvx_dz(gidz+1,gidx)-d_mem_dvx_dz(gidz-2,gidx)))/dz;
// 		dsxz_dz = (-c1*(d_sxz(gidz,gidx)-d_sxz(gidz-1,gidx)) + c2*(d_sxz(gidz+1,gidx)-d_sxz(gidz-2,gidx)))/dz;

// 		d_vx(gidz, gidx) += (d_a_x[gidx]*dpsixx_dx + lambda*dszz_dx/d_K_x[gidx]*dt \
// 				+ (lambda+2.0*mu)*dsxx_dx/d_K_x[gidx]*dt + d_a_z_half[gidz]*dpsixz_dz \
// 				+ d_ave_Mu(gidz,gidx)/d_K_z_half[gidz]*dsxz_dz*dt);

// 		//update phi_xx_x and phi_xz_z
// 		if(gidx<npml || gidx>nx-npml-1){
// 			d_mem_dsxx_dx(gidz, gidx) = d_b_x_half[gidx]*d_mem_dsxx_dx(gidz, gidx) + d_ave_Byc_b(gidz, gidx)*d_vx(gidz, gidx)*dt;
// 		}
// 		if(gidz<npml || (gidz>nz-npml-npad-1)){
// 			d_mem_dsxz_dz(gidz, gidx) = d_b_z[gidz]*d_mem_dsxz_dz(gidz, gidx) + d_ave_Byc_b(gidz, gidx)*d_vx(gidz, gidx)*dt;
// 		}

// 	  // update vz
// 		dpsizz_dz = (-c1*(d_mem_dvz_dz(gidz+1,gidx)-d_mem_dvz_dz(gidz,gidx)) \
// 			          + c2*(d_mem_dvz_dz(gidz+2,gidx)-d_mem_dvz_dz(gidz-1,gidx)))/dz;
// 		dszz_dz = (-c1*(d_szz(gidz+1,gidx)-d_szz(gidz,gidx)) + c2*(d_szz(gidz+2,gidx)-d_szz(gidz-1,gidx)))/dz;
// 		dsxx_dz = (-c1*(d_sxx(gidz+1,gidx)-d_sxx(gidz,gidx)) + c2*(d_sxx(gidz+2,gidx)-d_sxx(gidz-1,gidx)))/dz;
// 		dpsizx_dx = (-c1*(d_mem_dvz_dx(gidz,gidx)-d_mem_dvz_dx(gidz,gidx-1)) \
// 			           +c2*(d_mem_dvz_dx(gidz,gidx+1)-d_mem_dvz_dx(gidz,gidx-2)))/dx;
// 		dsxz_dx = (-c1*(d_sxz(gidz,gidx)-d_sxz(gidz,gidx-1)) + c2*(d_sxz(gidz,gidx+1)-d_sxz(gidz,gidx-2)))/dx;

// 		d_vz(gidz, gidx) += (d_a_z[gidz]*dpsizz_dz + (lambda+2.0*mu)*dszz_dz/d_K_z[gidz]*dt \
// 			+ lambda*dsxx_dz/d_K_z[gidz]*dt + d_a_x_half[gidx]*dpsizx_dx \
// 			+ d_ave_Mu(gidz,gidx)/d_K_x_half[gidx]*dsxz_dx*dt);

// 		// update phi_xz_x and phi_zz_z
// 		if(gidx<npml || gidx>nx-npml-1){
// 			d_mem_dsxz_dx(gidz, gidx) = d_b_x[gidx]*d_mem_dsxz_dx(gidz, gidx) + d_ave_Byc_a(gidz, gidx)*d_vz(gidz, gidx)*dt;
// 		}
// 		if(gidz<npml || (gidz>nz-npml-npad-1)){
// 			d_mem_dszz_dz(gidz, gidx) = d_b_z_half[gidz]*d_mem_dszz_dz(gidz, gidx) + d_ave_Byc_a(gidz, gidx)*d_vz(gidz, gidx)*dt;
// 		}

//   }

//   else {
//     return;
//   }
// }

// // elastic adjoint modeling: update stress
// __global__ void elastic_adjoint_stress(
//     float *d_vz, float *d_vx, float *d_szz, float *d_sxx, float *d_sxz,
//     float *d_mem_dszz_dz, float *d_mem_dsxz_dx, float *d_mem_dsxz_dz,
//     float *d_mem_dsxx_dx, float *d_mem_dvz_dz, float *d_mem_dvz_dx,
//     float *d_mem_dvx_dz, float *d_mem_dvx_dx, float *d_Lambda, float *d_Mu,
//     float *d_ave_Mu, float *d_Den, float *d_ave_Byc_a, float *d_ave_Byc_b,
//     float *d_K_z_half, float *d_a_z_half, float *d_b_z_half, float *d_K_x_half,
//     float *d_a_x_half, float *d_b_x_half, float *d_K_z, float *d_a_z,
//     float *d_b_z, float *d_K_x, float *d_a_x, float *d_b_x, int nz, int nx,
//     float dt, float dz, float dx, int npml, int npad) {

//   int gidz = blockIdx.x * blockDim.x + threadIdx.x;
//   int gidx = blockIdx.y * blockDim.y + threadIdx.y;

//   float dphi_xz_x_dx = 0.0;
//   float dvz_dx = 0.0;
//   float dphi_xz_z_dz = 0.0;
//   float dvx_dz = 0.0;
//   float dphi_xx_x_dx = 0.0;
//   float dvx_dx = 0.0;
//   float dphi_zz_z_dz = 0.0;
//   float dvz_dz = 0.0;

//   float c1 = 9.0 / 8.0;
//   float c2 = 1.0 / 24.0;

//   float lambda = d_Lambda(gidz, gidx);
//   float mu = d_Mu(gidz, gidx);

//   if (gidz >= 2 && gidz <= nz - npad - 3 && gidx >= 2 && gidx <= nx - 3) {

//  		dphi_xz_x_dx = (-c1*(d_mem_dsxz_dx(gidz,gidx+1)-d_mem_dsxz_dx(gidz,gidx)) \
// 				            +c2*(d_mem_dsxz_dx(gidz,gidx+2)-d_mem_dsxz_dx(gidz,gidx-1)))/dx;
// 		dvz_dx = (-c1*(d_vz(gidz,gidx+1)-d_vz(gidz,gidx)) + c2*(d_vz(gidz,gidx+2)-d_vz(gidz,gidx-1)))/dx;
// 		dphi_xz_z_dz = (-c1*(d_mem_dsxz_dz(gidz+1,gidx)-d_mem_dsxz_dz(gidz,gidx)) \
// 				            +c2*(d_mem_dsxz_dz(gidz+2,gidx)-d_mem_dsxz_dz(gidz-1,gidx)))/dz;
// 		dvx_dz = (-c1*(d_vx(gidz+1,gidx)-d_vx(gidz,gidx)) + c2*(d_vx(gidz+2,gidx)-d_vx(gidz-1,gidx)))/dz;

// 		// update sxz
// 		d_sxz(gidz,gidx) += d_a_x[gidx]*dphi_xz_x_dx + dvz_dx/d_K_x[gidx]*d_ave_Byc_a(gidz,gidx)*dt \
// 				              + d_a_z[gidz]*dphi_xz_z_dz + dvx_dz/d_K_z[gidz]*d_ave_Byc_b(gidz,gidx)*dt;

// 		// update psi_zx and psi_xz
// 		if(gidx<npml || gidx>nx-npml-1){
// 			d_mem_dvz_dx(gidz,gidx) = d_b_x_half[gidx]*d_mem_dvz_dx(gidz,gidx) + d_sxz(gidz,gidx)*d_ave_Mu(gidz,gidx)*dt;
// 		}
// 		if(gidz<npml || gidz>nz-npml-npad-1){
// 			d_mem_dvx_dz(gidz,gidx) = d_b_z_half[gidz]*d_mem_dvx_dz(gidz,gidx) + d_sxz(gidz,gidx)*d_ave_Mu(gidz,gidx)*dt;
// 		}
		  
// 		dphi_xx_x_dx = (-c1*(d_mem_dsxx_dx(gidz,gidx)-d_mem_dsxx_dx(gidz,gidx-1)) \
// 				            +c2*(d_mem_dsxx_dx(gidz,gidx+1)-d_mem_dsxx_dx(gidz,gidx-2)))/dx;
// 		dvx_dx = (-c1*(d_vx(gidz,gidx)-d_vx(gidz,gidx-1)) + c2*(d_vx(gidz,gidx+1)-d_vx(gidz,gidx-2)))/dx;
// 		dphi_zz_z_dz = (-c1*(d_mem_dszz_dz(gidz,gidx)-d_mem_dszz_dz(gidz-1,gidx)) \
// 				            +c2*(d_mem_dszz_dz(gidz+1,gidx)-d_mem_dszz_dz(gidz-2,gidx)))/dz;
// 		dvz_dz = (-c1*(d_vz(gidz,gidx)-d_vz(gidz-1,gidx)) + c2*(d_vz(gidz+1,gidx)-d_vz(gidz-2,gidx)))/dz;

// 		// update sxx and szz
// 		d_sxx(gidz,gidx) += d_a_x_half[gidx]*dphi_xx_x_dx	+ d_ave_Byc_b(gidz, gidx)*dvx_dx/d_K_x_half[gidx]*dt;;
// 		d_szz(gidz,gidx) += d_a_z_half[gidz]*dphi_zz_z_dz + d_ave_Byc_a(gidz, gidx)*dvz_dz/d_K_z_half[gidz]*dt;

// 		// update psi_xx and psi_zz
// 		if(gidx<npml || gidx>nx-npml-1){
// 			d_mem_dvx_dx(gidz, gidx) = d_b_x[gidx]*d_mem_dvx_dx(gidz, gidx) + lambda*d_szz(gidz, gidx)*dt \
// 				+ (lambda+2.0*mu)*d_sxx(gidz,gidx)*dt;
// 		}
// 		if(gidz<npml || (gidz>nz-npml-npad-1)){
// 			d_mem_dvz_dz(gidz, gidx) = d_b_z[gidz]*d_mem_dvz_dz(gidz, gidx) + (lambda+2.0*mu)*d_szz(gidz, gidx)*dt \
// 				+ lambda*d_sxx(gidz,gidx)*dt;
// 		}

//   } else {
//     return;
//   }
// }




// elastic adjoint modeling: update velocity
// __global__ void elastic_adjoint_velocity(
//     float *d_vz, float *d_vx, float *d_szz, float *d_sxx, float *d_sxz,
//     float *d_mem_dszz_dz, float *d_mem_dsxz_dx, float *d_mem_dsxz_dz,
//     float *d_mem_dsxx_dx, float *d_mem_dvz_dz, float *d_mem_dvz_dx,
//     float *d_mem_dvx_dz, float *d_mem_dvx_dx, float *d_Lambda, float *d_Mu,
//     float *d_ave_Mu, float *d_Den, float *d_ave_Byc_a, float *d_ave_Byc_b,
//     float *d_K_z_half, float *d_a_z_half, float *d_b_z_half, float *d_K_x_half,
//     float *d_a_x_half, float *d_b_x_half, float *d_K_z, float *d_a_z,
//     float *d_b_z, float *d_K_x, float *d_a_x, float *d_b_x, int nz, int nx,
//     float dt, float dz, float dx, int npml, int npad) {

//   int gidz = blockIdx.x * blockDim.x + threadIdx.x;
//   int gidx = blockIdx.y * blockDim.y + threadIdx.y;

//   float dpsixx_dx = 0.0;
//   float dszz_dx = 0.0;
//   float dsxx_dx = 0.0;
//   float dpsixz_dz = 0.0;
//   float dsxz_dz = 0.0;
//   float dpsizz_dz = 0.0;
//   float dszz_dz = 0.0;
//   float dsxx_dz = 0.0;
//   float dpsizx_dx = 0.0;
//   float dsxz_dx = 0.0;

//   float c1 = 9.0 / 8.0;
//   float c2 = 1.0 / 24.0;

//   float lambda = d_Lambda(gidz, gidx);
//   float mu = d_Mu(gidz, gidx);

//   if (gidz >= 2 && gidz <= nz - npad - 3 && gidx >= 2 && gidx <= nx - 3) {

// 		// update vx
// 		dszz_dx = (-c1*(d_szz(gidz,gidx+1)-d_szz(gidz,gidx)) + c2*(d_szz(gidz,gidx+2)-d_szz(gidz,gidx-1)))/dx;
// 		dsxx_dx = (-c1*(d_sxx(gidz,gidx+1)-d_sxx(gidz,gidx)) + c2*(d_sxx(gidz,gidx+2)-d_sxx(gidz,gidx-1)))/dx;
// 		dsxz_dz = (-c1*(d_sxz(gidz,gidx)-d_sxz(gidz-1,gidx)) + c2*(d_sxz(gidz+1,gidx)-d_sxz(gidz-2,gidx)))/dz;

//     // pml boundary
//     // if (gidx < npml || gidx > nx - npml) {
//     //   d_mem_dsxx_dx(gidz, gidx) = d_b_x_half[gidx] * d_mem_dsxx_dx(gidz, gidx) + d_a_x_half[gidx] * dsxx_dx;

//     //   dsxx_dx = dsxx_dx / d_K_x_half[gidx] + d_mem_dsxx_dx(gidz, gidx);
//     //   dszz_dx = dszz_dx / d_K_x_half[gidx] + d_mem_dsxx_dx(gidz, gidx);
//     // }

//     // // pml boundary
//     // if (gidz < npml || (gidz > nz - npml - npad - 1)) {
//     //   d_mem_dsxz_dz(gidz, gidx) = d_b_z[gidz] * d_mem_dsxz_dz(gidz, gidx) + d_a_z[gidz] * dsxz_dz;
//     //   dsxz_dz = dsxz_dz / d_K_z[gidz] + d_mem_dsxz_dz(gidz, gidx);
//     // }

// 		d_vx(gidz, gidx) += ((lambda+2.0*mu)*dsxx_dx + lambda * dszz_dx + d_ave_Mu(gidz,gidx) * dsxz_dz) *dt;
    

// 	  // update vz
// 		dszz_dz = (-c1*(d_szz(gidz+1,gidx)-d_szz(gidz,gidx)) + c2*(d_szz(gidz+2,gidx)-d_szz(gidz-1,gidx)))/dz;
// 		dsxx_dz = (-c1*(d_sxx(gidz+1,gidx)-d_sxx(gidz,gidx)) + c2*(d_sxx(gidz+2,gidx)-d_sxx(gidz-1,gidx)))/dz;
// 		dsxz_dx = (-c1*(d_sxz(gidz,gidx)-d_sxz(gidz,gidx-1)) + c2*(d_sxz(gidz,gidx+1)-d_sxz(gidz,gidx-2)))/dx;

//     // // pml boundary
//     // if (gidz < npml || (gidz > nz - npml - npad - 1)) {
//     //   d_mem_dszz_dz(gidz, gidx) = d_b_z_half[gidz] * d_mem_dszz_dz(gidz, gidx) + d_a_z_half[gidz] * dszz_dz;

//     //   dszz_dz = dszz_dz / d_K_z_half[gidz] + d_mem_dszz_dz(gidz, gidx);
//     //   dsxx_dz = dsxx_dz / d_K_z_half[gidz] + d_mem_dszz_dz(gidz, gidx);
//     // }

//     // // pml boundary
//     // if (gidx < npml || gidx > nx - npml) {
//     //   d_mem_dsxz_dx(gidz, gidx) = d_b_x[gidx] * d_mem_dsxz_dx(gidz, gidx) + d_a_x[gidx] * dsxz_dx;
//     //   dsxz_dx = dsxz_dx / d_K_x[gidx] + d_mem_dsxz_dx(gidz, gidx);
//     // }
  
//     d_vz(gidz, gidx) += ((lambda+2.0*mu)*dszz_dz + lambda * dsxx_dz + d_ave_Mu(gidz,gidx) * dsxz_dx) *dt;

//   }

//   else {
//     return;
//   }
// }

// // elastic adjoint modeling: update stress
// __global__ void elastic_adjoint_stress(
//     float *d_vz, float *d_vx, float *d_szz, float *d_sxx, float *d_sxz,
//     float *d_mem_dszz_dz, float *d_mem_dsxz_dx, float *d_mem_dsxz_dz,
//     float *d_mem_dsxx_dx, float *d_mem_dvz_dz, float *d_mem_dvz_dx,
//     float *d_mem_dvx_dz, float *d_mem_dvx_dx, float *d_Lambda, float *d_Mu,
//     float *d_ave_Mu, float *d_Den, float *d_ave_Byc_a, float *d_ave_Byc_b,
//     float *d_K_z_half, float *d_a_z_half, float *d_b_z_half, float *d_K_x_half,
//     float *d_a_x_half, float *d_b_x_half, float *d_K_z, float *d_a_z,
//     float *d_b_z, float *d_K_x, float *d_a_x, float *d_b_x, int nz, int nx,
//     float dt, float dz, float dx, int npml, int npad) {

//   int gidz = blockIdx.x * blockDim.x + threadIdx.x;
//   int gidx = blockIdx.y * blockDim.y + threadIdx.y;

//   float dphi_xz_x_dx = 0.0;
//   float dvz_dx = 0.0;
//   float dphi_xz_z_dz = 0.0;
//   float dvx_dz = 0.0;
//   float dphi_xx_x_dx = 0.0;
//   float dvx_dx = 0.0;
//   float dphi_zz_z_dz = 0.0;
//   float dvz_dz = 0.0;

//   float c1 = 9.0 / 8.0;
//   float c2 = 1.0 / 24.0;

//   float lambda = d_Lambda(gidz, gidx);
//   float mu = d_Mu(gidz, gidx);

//   if (gidz >= 2 && gidz <= nz - npad - 3 && gidx >= 2 && gidx <= nx - 3) {


//   	// update sxz
// 		dvz_dx = (-c1*(d_vz(gidz,gidx+1)-d_vz(gidz,gidx)) + c2*(d_vz(gidz,gidx+2)-d_vz(gidz,gidx-1)))/dx;
// 		dvx_dz = (-c1*(d_vx(gidz+1,gidx)-d_vx(gidz,gidx)) + c2*(d_vx(gidz+2,gidx)-d_vx(gidz-1,gidx)))/dz;

//     // if (gidz < npml || (gidz > nz - npml - npad - 1)) {
//     //   d_mem_dvx_dz(gidz, gidx) = d_b_z_half[gidz] * d_mem_dvx_dz(gidz, gidx) + d_a_z_half[gidz] * dvx_dz;
//     //   dvx_dz = dvx_dz / d_K_z_half[gidz] + d_mem_dvx_dz(gidz, gidx);
//     // }
//     // if (gidx < npml || gidx > nx - npml - 1) {
//     //   d_mem_dvz_dx(gidz, gidx) = d_b_x_half[gidx] * d_mem_dvz_dx(gidz, gidx) + d_a_x_half[gidx] * dvz_dx;
//     //   dvz_dx = dvz_dx / d_K_x_half[gidx] + d_mem_dvz_dx(gidz, gidx);
//     // }

// 		// update sxz
// 		d_sxz(gidz,gidx) +=  dvz_dx * d_ave_Byc_a(gidz,gidx)*dt + dvx_dz * d_ave_Byc_b(gidz,gidx)*dt;

//     // update sxx and szz
// 		dvx_dx = (-c1*(d_vx(gidz,gidx)-d_vx(gidz,gidx-1)) + c2*(d_vx(gidz,gidx+1)-d_vx(gidz,gidx-2)))/dx;
// 		dvz_dz = (-c1*(d_vz(gidz,gidx)-d_vz(gidz-1,gidx)) + c2*(d_vz(gidz+1,gidx)-d_vz(gidz-2,gidx)))/dz;

//     // if (gidz < npml || (gidz > nz - npml - npad - 1)) {
//     //   d_mem_dvz_dz(gidz, gidx) = d_b_z[gidz] * d_mem_dvz_dz(gidz, gidx) + d_a_z[gidz] * dvz_dz;
//     //   dvz_dz = dvz_dz / d_K_z[gidz] + d_mem_dvz_dz(gidz, gidx);
//     // }
//     // if (gidx < npml || gidx > nx - npml - 1) {
//     //   d_mem_dvx_dx(gidz, gidx) = d_b_x[gidx] * d_mem_dvx_dx(gidz, gidx) + d_a_x[gidx] * dvx_dx;
//     //   dvx_dx = dvx_dx / d_K_x[gidx] + d_mem_dvx_dx(gidz, gidx);
//     // }

// 		// update sxx and szz
// 		d_sxx(gidz,gidx) += d_ave_Byc_b(gidz, gidx) * dvx_dx * dt;
// 		d_szz(gidz,gidx) += d_ave_Byc_a(gidz, gidx) * dvz_dz * dt;

//   } else {
//     return;
//   }
// }