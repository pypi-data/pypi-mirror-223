#include "Boundary.h"
#include "Parameter.h"
#include "utils.h"

Bnd::Bnd(const Parameter &para, bool const with_grad) {
  with_grad_ = with_grad;
  if (with_grad_) {
    nz_ = para.nz();
    nx_ = para.nx();
    npml_ = para.npml();
    npad_ = para.npad();
    nt_ = para.nt();

    // save extra 2 layers in the pml for derivative at the boundaries
    nzBnd_ = nz_ - 2 * npml_ - npad_ + 4;
    nxBnd_ = nx_ - 2 * npml_ + 4;
    nLayerStore_ = 5;

    // store n layers
    len_Bnd_vec_ = 2 * (nLayerStore_ * nzBnd_ + nLayerStore_ * nxBnd_);

    // allocate the boundary vector in the device
    CHECK(hipMalloc((void **)&d_Bnd_szz, len_Bnd_vec_ * nt_ * sizeof(float)));
    CHECK(hipMalloc((void **)&d_Bnd_sxz, len_Bnd_vec_ * nt_ * sizeof(float)));
    CHECK(hipMalloc((void **)&d_Bnd_sxx, len_Bnd_vec_ * nt_ * sizeof(float)));

    CHECK(hipMalloc((void **)&d_Bnd_vz, len_Bnd_vec_ * nt_ * sizeof(float)));
    CHECK(hipMalloc((void **)&d_Bnd_vx, len_Bnd_vec_ * nt_ * sizeof(float)));
  }
}

Bnd::~Bnd() {
  if (with_grad_) {
    CHECK(hipFree(d_Bnd_szz));
    CHECK(hipFree(d_Bnd_sxz));
    CHECK(hipFree(d_Bnd_sxx));
    CHECK(hipFree(d_Bnd_vz));
    CHECK(hipFree(d_Bnd_vx));
  }
}

void Bnd::field_from_bnd(float *d_szz, float *d_sxz, float *d_sxx, float *d_vz,
                         float *d_vx, int indT) {
  from_bnd<<<(len_Bnd_vec_ + 31) / 32, 32>>>(d_szz, d_Bnd_szz, nz_, nx_, nzBnd_,
                                             nxBnd_, len_Bnd_vec_, nLayerStore_,
                                             indT, npml_, npad_, nt_);

  from_bnd<<<(len_Bnd_vec_ + 31) / 32, 32>>>(d_sxz, d_Bnd_sxz, nz_, nx_, nzBnd_,
                                             nxBnd_, len_Bnd_vec_, nLayerStore_,
                                             indT, npml_, npad_, nt_);

  from_bnd<<<(len_Bnd_vec_ + 31) / 32, 32>>>(d_sxx, d_Bnd_sxx, nz_, nx_, nzBnd_,
                                             nxBnd_, len_Bnd_vec_, nLayerStore_,
                                             indT, npml_, npad_, nt_);

  from_bnd<<<(len_Bnd_vec_ + 31) / 32, 32>>>(d_vz, d_Bnd_vz, nz_, nx_, nzBnd_,
                                             nxBnd_, len_Bnd_vec_, nLayerStore_,
                                             indT, npml_, npad_, nt_);

  from_bnd<<<(len_Bnd_vec_ + 31) / 32, 32>>>(d_vx, d_Bnd_vx, nz_, nx_, nzBnd_,
                                             nxBnd_, len_Bnd_vec_, nLayerStore_,
                                             indT, npml_, npad_, nt_);
}

void Bnd::field_to_bnd(float *d_szz, float *d_sxz, float *d_sxx, float *d_vz,
                       float *d_vx, int indT, bool if_stress) {
  if (if_stress) {
    to_bnd<<<(len_Bnd_vec_ + 31) / 32, 32>>>(d_szz, d_Bnd_szz, nz_, nx_, nzBnd_,
                                             nxBnd_, len_Bnd_vec_, nLayerStore_,
                                             indT, npml_, npad_, nt_);

    to_bnd<<<(len_Bnd_vec_ + 31) / 32, 32>>>(d_sxz, d_Bnd_sxz, nz_, nx_, nzBnd_,
                                             nxBnd_, len_Bnd_vec_, nLayerStore_,
                                             indT, npml_, npad_, nt_);

    to_bnd<<<(len_Bnd_vec_ + 31) / 32, 32>>>(d_sxx, d_Bnd_sxx, nz_, nx_, nzBnd_,
                                             nxBnd_, len_Bnd_vec_, nLayerStore_,
                                             indT, npml_, npad_, nt_);

  } else {
    to_bnd<<<(len_Bnd_vec_ + 31) / 32, 32>>>(d_vz, d_Bnd_vz, nz_, nx_, nzBnd_,
                                             nxBnd_, len_Bnd_vec_, nLayerStore_,
                                             indT, npml_, npad_, nt_);

    to_bnd<<<(len_Bnd_vec_ + 31) / 32, 32>>>(d_vx, d_Bnd_vx, nz_, nx_, nzBnd_,
                                             nxBnd_, len_Bnd_vec_, nLayerStore_,
                                             indT, npml_, npad_, nt_);
  }
}